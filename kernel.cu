#include "hip/hip_runtime.h"
#include "kernel.h"

#define BASE 262144 //1024*1024

#define block_size 512	
#define thread_size 512
#define block_size2 16
#define thread_size2 512
#define WARP_SIZE 32


/*the device function for level 2 reduce*/
__device__ void segreduce_block(const int * idx, double * val)
{
    double left = 0;
    if( threadIdx.x >=   1 && idx[threadIdx.x] == idx[threadIdx.x -   1] ) { left = val[threadIdx.x -   1]; } __syncthreads(); val[threadIdx.x] += left; left = 0; __syncthreads();  
    if( threadIdx.x >=   2 && idx[threadIdx.x] == idx[threadIdx.x -   2] ) { left = val[threadIdx.x -   2]; } __syncthreads(); val[threadIdx.x] += left; left = 0; __syncthreads();
    if( threadIdx.x >=   4 && idx[threadIdx.x] == idx[threadIdx.x -   4] ) { left = val[threadIdx.x -   4]; } __syncthreads(); val[threadIdx.x] += left; left = 0; __syncthreads();
    if( threadIdx.x >=   8 && idx[threadIdx.x] == idx[threadIdx.x -   8] ) { left = val[threadIdx.x -   8]; } __syncthreads(); val[threadIdx.x] += left; left = 0; __syncthreads();
    if( threadIdx.x >=  16 && idx[threadIdx.x] == idx[threadIdx.x -  16] ) { left = val[threadIdx.x -  16]; } __syncthreads(); val[threadIdx.x] += left; left = 0; __syncthreads();
    if( threadIdx.x >=  32 && idx[threadIdx.x] == idx[threadIdx.x -  32] ) { left = val[threadIdx.x -  32]; } __syncthreads(); val[threadIdx.x] += left; left = 0; __syncthreads();  
    if( threadIdx.x >=  64 && idx[threadIdx.x] == idx[threadIdx.x -  64] ) { left = val[threadIdx.x -  64]; } __syncthreads(); val[threadIdx.x] += left; left = 0; __syncthreads();
    if( threadIdx.x >= 128 && idx[threadIdx.x] == idx[threadIdx.x - 128] ) { left = val[threadIdx.x - 128]; } __syncthreads(); val[threadIdx.x] += left; left = 0; __syncthreads();
    if( threadIdx.x >= 256 && idx[threadIdx.x] == idx[threadIdx.x - 256] ) { left = val[threadIdx.x - 256]; } __syncthreads(); val[threadIdx.x] += left; left = 0; __syncthreads();
	if( threadIdx.x >= 512 && idx[threadIdx.x] == idx[threadIdx.x - 512] ) { left = val[threadIdx.x - 512]; } __syncthreads(); val[threadIdx.x] += left; left = 0; __syncthreads();
}

__device__ 
double get_val(const unsigned int idx, const unsigned int scope1, const unsigned int scope2,  const double *vec, volatile double* cached_vec){
	if(idx > scope1 && idx < scope2)
		return cached_vec[idx - scope1];
	else
		return vec[idx];
}

/*kernel function for initialize*/
__global__ void kernelInitialize(const unsigned int num, double *x)
{
	unsigned int idx=blockDim.x * blockIdx.x+ threadIdx.x;
	
	for (unsigned int n=idx;n<num;n+=BASE) x[n]=0;
}

__global__ void kernelInitializeAll(const unsigned int num, double *pk, double *bp, double *x, double *zk, const double *vector_in)
{
	unsigned int idx=blockDim.x * blockIdx.x+ threadIdx.x;
	double temp;
	for (unsigned int n=idx;n<num;n+=BASE) 
	{
		temp=zk[n];
		pk[n]=temp;
		bp[n]=0;
		x[n]=0;
	}
}

__global__ void kernelInitializeR(const unsigned int num,double *rk, const double *vector_in)
{
	unsigned int idx=blockDim.x * blockIdx.x+ threadIdx.x;
	double temp;
	for (unsigned int n=idx;n<num;n+=BASE) 
	{
		temp=vector_in[n];
		rk[n]=temp;
	}
}

//for ELL format matrix, output y=data*x
__global__ void ELL_kernel(const unsigned int num_rows, const unsigned int cal_rows, const unsigned int num_cols_per_row,
			const unsigned int *indices, const double *data, const double * x, double * y, 
			const unsigned int bias0, const unsigned int bias1)
{
	unsigned int row= blockDim.x*blockIdx.x+threadIdx.x;
	if (row<cal_rows){
		double dot =0;
		for (unsigned int n=0; n< num_cols_per_row; n++){
			unsigned int col=indices[num_rows * n + row];
			double val=data[num_rows*n+row];

			if (val != 0)
				dot += val* x[col-bias0];
		}
		y[row+bias1]+=dot;
	}
}
/*
__global__ void ELL_kernel_float(const unsigned int num_rows, const unsigned int cal_rows, const unsigned int num_cols_per_row,
			const unsigned int *indices, const float* data, const double * x, double * y, 
			const unsigned int bias0, const unsigned int bias1)
{
	unsigned int row= blockDim.x*blockIdx.x+threadIdx.x;
	if (row<cal_rows){
		double dot =0;
		for (unsigned int n=0; n< num_cols_per_row; n++){
			unsigned int col=indices[num_rows * n + row];
			double val=data[num_rows*n+row];

			if (val != 0)
				dot += val* x[col-bias0];
		}
		y[row+bias1]+=dot;
	}
}*/
__global__ void ELL_kernel_block(const unsigned int num_rows, const unsigned int cal_rows, 
			const unsigned int* num_cols_per_row_vec, const unsigned int* block_data_bias_vec,  
			const unsigned int *indices, const double *data, const double * x, double * y, 
			const unsigned int bias0, const unsigned int bias1){
	
	unsigned int block_idx = blockIdx.x; 
	unsigned int thread_idx = threadIdx.x; 
	unsigned int num_cols_per_row;  
	unsigned int block_data_bias;
	unsigned int data;
	unsigned int col;	
	double val;
	unsigned int row= blockDim.x*blockIdx.x+threadIdx.x;
	if(row < cal_rows){
		num_cols_per_row = num_cols_per_row_vec[block_idx];//cache will work when every threads read same global address
		block_data_bias = block_data_bias_vec[block_idx];
		double dot =0;
		for (unsigned int n=0; n< num_cols_per_row; n++){
			data_idx = block_data_bias + ELL_threadSize*n + thread_idx;
			col=indices[data_idx];
			val=data[data_idx];

			if (val != 0)
				dot += val* x[col-bias0];
		}
		y[row+bias1]+=dot;
	}
}

/* bias0 and bias1 is reserved for future distributed version*/
__global__ void ELL_cached_kernel(const unsigned int num_rows,  
				const unsigned int num_cols_per_row, 
				const unsigned int *indices, const double *data, const double * x,
				double * y, const unsigned int bias0, 
				const unsigned int bias1, const unsigned int* part_boundary)
{
	unsigned int x_idx = blockDim.x*blockIdx.x+threadIdx.x;
	__shared__ volatile double cached_vec[vector_cache_size];  
	unsigned int vec_start = part_boundary[blockIdx.x] + bias0;
	unsigned int vec_end = part_boundary[blockIdx.x + 1] + bias0;
	double val, dot;
	unsigned int col;

	for (unsigned int i = x_idx; i < vector_cache_size; i += ELL_threadSize){
		if(i < vec_end) cached_vec[i] = x[i + vec_start];
		else cached_vec[i] = 0;
	}
	for(unsigned int row = x_idx; row < vec_end; row += ELL_threadSize){
		dot =0;
		for (unsigned int n=0; n< num_cols_per_row; n++){
			col=indices[num_rows*n + row];
			val=data[num_rows*n + row];
			if (val != 0)
				dot += val*get_val(col, vec_start, vec_start + vector_cache_size, x, cached_vec);
		}
		y[row+bias1] = dot;
	}		
}

__global__ void ELL_cached_kernel_block(const unsigned int* num_cols_per_row_vec, 
		const unsigned int* block_data_bias_vec,  
		const unsigned int *indices, const double *data, const double * x,
		double * y, const unsigned int bias0, 
		const unsigned int bias1, const unsigned int* part_boundary){

	unsigned int block_idx = blockIdx.x; 
	unsigned int x_idx = blockDim.x*blockIdx.x+threadIdx.x;
	__shared__ volatile double cached_vec[vector_cache_size];  
	unsigned int vec_start = part_boundary[blockIdx.x] + bias0;
	unsigned int vec_end = part_boundary[blockIdx.x + 1] + bias0;
	double val, dot;
	unsigned int col;
	num_cols_per_row = num_cols_per_row_vec[block_idx];//cache will work when every threads read same global address
	//vec_start + vector_cache_size will be slightly different from vec_end
	for (unsigned int i = x_idx; i < vector_cache_size; i += ELL_threadSize){
		if(i < vec_end) cached_vec[i] = x[i + vec_start];
		else cached_vec[i] = 0;
	}

	block_data_bias = block_data_bias_vec[block_idx];
	for(unsigned int row = x_idx; row < vec_end; row += ELL_threadSize){
		dot =0;
		for (unsigned int n=0; n< num_cols_per_row; n++){
			data_idx = block_data_bias + ELL_threadSize*n + row;
			col=indices[data_idx];
			val=data[data_idx];
			if (val != 0)
				dot += val*get_val(col, vec_start, vec_start + vector_cache_size, x, cached_vec);
		}
		y[row+bias1] = dot;
	}		
}
/*
__global__ void ELL_cached_kernel_float(const unsigned int num_rows,  
				const unsigned int num_cols_per_row, 
				const unsigned int *indices, const float* data, const double * x,
				double * y, const unsigned int bias0, 
				const unsigned int bias1, const unsigned int* part_boundary)
{
	unsigned int x_idx = blockDim.x*blockIdx.x+threadIdx.x;
	__shared__ volatile double cached_vec[vector_cache_size];  
	unsigned int vec_start = part_boundary[blockIdx.x] + bias0;
	unsigned int vec_end = part_boundary[blockIdx.x + 1] + bias0;
	double val, dot;
	unsigned int col;

	for (unsigned int i = x_idx; i < vector_cache_size; i += ELL_threadSize){
		cached_vec[i] = x[i + vec_start];
	}
	for(unsigned int row = x_idx; row < vec_end; row += ELL_threadSize){
		dot =0;
		for (unsigned int n=0; n< num_cols_per_row; n++){
			col=indices[num_rows*n + row];
			val=data[num_rows*n + row];
			if (val != 0)
				dot += val*get_val(col, vec_start, vec_start + vector_cache_size, x, cached_vec);
		}
		y[row+bias1] = dot;
	}		
}*/

//for COO format matrix, output y=data*x
//the basic idea is come from
__global__ void COO_level1(const unsigned int num_nozeros, const unsigned int interval_size, 
				const unsigned int *I, const unsigned int *J, const double *V, 
				const double *x, double *y, int *temp_rows, 
				double *temp_vals, const unsigned int xp,const unsigned int yp)
{
	__shared__ volatile int rows[48*thread_size/WARP_SIZE];  //why using 48? because we need 16 additional junk elements
	__shared__ volatile double vals[thread_size];
	
	unsigned int thread_id = blockDim.x*blockIdx.x + threadIdx.x;
	unsigned int thread_lane= threadIdx.x & (WARP_SIZE-1); //great idea! think about it
	unsigned int warp_id = thread_id / WARP_SIZE;
	
	unsigned int interval_begin=warp_id*interval_size;
	unsigned int interval_end =min(interval_begin+interval_size, num_nozeros);
	/*how about the interval is not the multiple of warp_size?*/
	//unsigned int iteration_end=((interval_end)/WARP_SIZE)*WARP_SIZE;
	
	unsigned int idx=16*(threadIdx.x/32+1) + threadIdx.x;//every warp has 16 "junk" rows elements
	
	rows[idx-16]=-1;
	
	if(interval_begin >= interval_end)
	{
		temp_rows[warp_id] = -1;
		return;
	}	
	if (thread_lane ==31)
	{
		// initialize the cary in values
		rows[idx]=I[interval_begin];
		vals[threadIdx.x]=0;
	}
	unsigned int n;
	n=interval_begin+thread_lane;
	while (n< interval_end)
	{
		unsigned int row =I[n];
		//double val=V[n]*fetch_x(J[n], x);
		double val=V[n]*x[J[n]-xp];
		
		if (thread_lane==0)
		{
			if (row==rows[idx+31])
				val+=vals[threadIdx.x+31]; //don't confused by the "plus" 31, because the former end is the new start
			else 
				y[rows[idx+31]-yp] += vals[threadIdx.x+31];//try to fix the bug from orignial library functions
		}
		rows[idx] =row;
		vals[threadIdx.x] =val;
		
        if(row == rows[idx -  1]) { vals[threadIdx.x] = val = val + vals[threadIdx.x -  1]; } 
        if(row == rows[idx -  2]) { vals[threadIdx.x] = val = val + vals[threadIdx.x -  2]; }
        if(row == rows[idx -  4]) { vals[threadIdx.x] = val = val + vals[threadIdx.x -  4]; }
        if(row == rows[idx -  8]) { vals[threadIdx.x] = val = val + vals[threadIdx.x -  8]; }
        if(row == rows[idx - 16]) { vals[threadIdx.x] = val = val + vals[threadIdx.x - 16]; }

        if(thread_lane < 31 && row < rows[idx + 1] && n<interval_end-1)
            y[row-yp] += vals[threadIdx.x];  
		n+=WARP_SIZE;
	}
	
	/*now we consider the reminder of interval_size/warp_size*/

    /*program at one warp is automatically sychronized*/
	if(n==(interval_end+WARP_SIZE-1))
    {
        // write the carry out values
        temp_rows[warp_id] = rows[idx];
        temp_vals[warp_id] = vals[threadIdx.x];
    }	
	
}

/* The second level of the segmented reduction operation
Why we need second level of reduction? because the program running at different block can not be sychronized 
Notice the number of input elements is fixed, and the number is relatively much small than the dimension of matrixs
consider block_size=512, thread_size=512 (i.e. 512 block, each block has 512 threads) and wrapSize=32 the dimension of
temp_rows will be 512*512/32=8192, this is a fix number
So, we should set this device function's block_size=512/32=16, thread_size=512*/

__global__ void COO_level2(const int * temp_rows,
                       	const double * temp_vals,
			int * temp_rows2,
			double * temp_vals2,
                       	double * y, const unsigned int p)
/*The bias is */									
{
    __shared__ int rows[thread_size2 + 1];    
    __shared__ double vals[thread_size2 + 1];
	unsigned int idx_t=threadIdx.x;
	unsigned int idx_g=blockDim.x*blockIdx.x+threadIdx.x;
	
    if (threadIdx.x == 0)
    {
        rows[thread_size2] =  -1;
        vals[thread_size2] =   0;
	temp_rows2[blockIdx.x]=-1;
    }
    	
	rows[idx_t]=temp_rows[idx_g];
	vals[idx_t]=temp_vals[idx_g];
	__syncthreads();
	
	segreduce_block(rows, vals);
	
	if (rows[threadIdx.x] != rows[threadIdx.x + 1])
	{
		if (threadIdx.x!=(thread_size2-1))
		{
			if (rows[threadIdx.x]>=0 && rows[threadIdx.x]>=p) y[rows[threadIdx.x]-p] += vals[threadIdx.x];
		}
		else
		{
			temp_rows2[blockIdx.x]=rows[threadIdx.x];
			temp_vals2[blockIdx.x]=vals[threadIdx.x];
		}
	}		
	
}

//no sychronize between blocks, so we need to restart another kernel function
__global__ void COO_level3(const unsigned int num,
                            const int * temp_rows,
                            double * temp_vals,
                            double * y,const unsigned int p)
{
	/*only 16 elements, single thread is enough*/
	unsigned int i=0;
	for (i=0;i<num-1;i++)
	{
		if (temp_rows[i]!=temp_rows[i+1])
			if (temp_rows[i]>=0 && temp_rows[i]>=p) y[temp_rows[i]-p] +=temp_vals[i];
		else if (temp_rows[i]==temp_rows[i+1])
			temp_vals[i+1]=temp_vals[i+1]+temp_vals[i]; //don't forget update the values! also at most situation (sparse matrix) it is unnecessary	
	}
	/*the last elements of input data will not disturb by any other elements, so update the output directly*/
	if (temp_rows[i]>=0 && temp_rows[i]>=p) 
		y[temp_rows[i]-p] +=temp_vals[i];
}

/*The single thread version of reduction*/
__global__ void COO_level2_serial(const int * temp_rows,
                              const double * temp_vals,
                                    double * y,const unsigned int p)
{
	unsigned int i=0;
	for (i=0;i<(block_size*thread_size/WARP_SIZE);i++)
	{
		if (temp_rows[i]>=0 && temp_rows[i]>=p) 
			y[temp_rows[i]-p]+=temp_vals[i];
	}
}

__global__ void COO_level2_serial2(const int * temp_rows,
                              const double * temp_vals,
                                    double * y, const unsigned int p)
{
	unsigned int i=0;
	for (i=0;i<(block_size2*thread_size2/WARP_SIZE);i++)
	{
		
	if (temp_rows[i]>=0 && temp_rows[i]>=p) 
 		y[temp_rows[i]-p]+=temp_vals[i];
	}
}

__global__ void COO_level2_serial3(const unsigned int num, const int * temp_rows,
                              const double * temp_vals,
                                    double * y,const unsigned int p)
{
	unsigned int i=0;
	for (i=0;i<num;i++)
	{
		
	if (temp_rows[i]>=0 && temp_rows[i]>=p) 
 		y[temp_rows[i]-p]+=temp_vals[i];
	}
}

__global__ void COO_level1_serial(const unsigned int num, unsigned int *I, unsigned int *J, double *V, double *x, double *y, const unsigned int xp, const unsigned int yp)
{
	unsigned int i=0;
	for (i=0;i<num;i++)
	{
		if (I[i]>=yp) y[I[i]-yp]+=V[i]*x[J[i]-xp];
	}
}

//y=x+gamak*y
__global__ void kernelMyxpy(const unsigned int dimension, double gamak, const double *x, double *y)
{
	unsigned int idx=blockDim.x*blockIdx.x+threadIdx.x;
	unsigned int n=idx;
	while(n<dimension){
		y[n]=x[n]+gamak*y[n];
		n=n+BASE;
	}
}
extern "C"
void initialize_all(const unsigned int dimension, double *pk_d, double *bp_d, double *x, double *zk, const double *vector_in_d)
{
	kernelInitializeAll<<<block_size,thread_size>>>(dimension, pk_d, bp_d, x, zk, vector_in_d);
}

void initialize_bp(unsigned int num, double *x)
{
	kernelInitialize<<<block_size,thread_size>>>(num,x);
}

void initialize_r(unsigned int num, double *rk, double *vector_in)
{
	kernelInitializeR<<<block_size,thread_size>>>(num,rk,vector_in);
}
void myxpy(const unsigned int dimension, double gamak, const double *x, double *y)
{
	kernelMyxpy<<<block_size,thread_size>>>(dimension,gamak,x,y);
}

void initialDeviceArray(unsigned int num, double *x)
{
	kernelInitialize<<<512,512>>>(num,x);
}


void matrix_vectorELL(const unsigned int num_rows, const unsigned int cal_rows, 
			const unsigned int num_cols_per_row,  const unsigned int *J,
 			const double *V, const double *x, double *y, const unsigned int bias0, const unsigned int bias1, 
			const bool RODR, const unsigned int rodr_blocks, const unsigned int* part_boundary_d)
{
	/*bias0 is for x and bias1 is for y, in precond solver, x, y may have different start point, 
		bias0 is "absolut bias", bias 1 is relative bias*/
	unsigned int ELL_blocks = ceil((double) num_rows/ELL_threadSize);
	//printf("ELL_blocks is %d\n", ELL_blocks);
	//bind_x(x);
	if(RODR){
		
		ELL_cached_kernel<<<rodr_blocks, ELL_threadSize>>>(num_rows, num_cols_per_row, J,
 			V, x,y, bias0, bias1, part_boundary_d);	
	} else { 
		ELL_kernel<<<ELL_blocks, ELL_threadSize>>>(num_rows, cal_rows, num_cols_per_row, J, V, x,y, bias0, bias1);
	}
	//unbind_x(x);
	
}

/*void matrix_vectorELL_float(const unsigned int num_rows, const unsigned int cal_rows, 
			const unsigned int num_cols_per_row,  const unsigned int *J,
 			const float* V, const double *x, double *y, const unsigned int bias0, const unsigned int bias1, 
			const bool RODR, const unsigned int rodr_blocks, const unsigned int* part_boundary_d)
{
	/*bias0 is for x and bias1 is for y, in precond solver, x, y may have different start point,
		bias0 is "absolut bias", bias 1 is relative bias*
	unsigned int ELL_blocks = ceil((double) num_rows/ELL_threadSize);
	//printf("ELL_blocks is %d\n", ELL_blocks);
	//bind_x(x);
	if(RODR){
		
		ELL_cached_kernel_float<<<rodr_blocks, ELL_threadSize>>>(num_rows, num_cols_per_row, J,
 			V, x,y, bias0, bias1, part_boundary_d);	
	} else { 
		ELL_kernel_float<<<ELL_blocks, ELL_threadSize>>>(num_rows, cal_rows, num_cols_per_row, J, V, x,y, bias0, bias1);
	}
	//unbind_x(x);
	
}*/

void matrix_vectorELL_block(const unsigned int num_rows, const unsigned int cal_rows, 
			const unsigned int* num_cols_per_row_vec, 
			const unsigned int* block_data_bias_vec,    
			const unsigned int *J,
 			const double *V, const double *x, double *y, const unsigned int bias0, const unsigned int bias1, 
			const bool RODR, const unsigned int rodr_blocks, const unsigned int* part_boundary_d)
{
	/*bias0 is for x and bias1 is for y, in precond solver, x, y may have different start point, 
		bias0 is "absolut bias", bias 1 is relative bias*/
	unsigned int ELL_blocks = ceil((double) num_rows/ELL_threadSize);
	//printf("ELL_blocks is %d\n", ELL_blocks);
	//bind_x(x);
	if(RODR){
		
		ELL_cached_kernel_block<<<rodr_blocks, ELL_threadSize>>>(num_rows, num_cols_per_row_vec, 
			block_data_bias_vec,
			J, V, x, y, bias0, bias1, part_boundary_d);
	}
	else
		ELL_kernel_block<<<ELL_blocks, ELL_threadSize>>>(num_rows, cal_rows, num_cols_per_row_vec, 
			block_data_bias_vec, J, V, x,y, bias0, bias1);
	//unbind_x(x);
	
}

void matrix_vectorCOO(const unsigned int num_nozeros_compensation, unsigned int *I, unsigned int *J, double *V, double *x_d, double *y_d, unsigned int bias0, unsigned int bias1)
{
	/*bias0 is for input vector, bias1 is for output vector, different from the ELL format both bias0 and bias1 is absolut bias*/
	unsigned int interval_size2;
	interval_size2=ceil(((double) num_nozeros_compensation)/(block_size*thread_size/WARP_SIZE));//for data with 2 million elements, we have interval size 200	
	//printf("num_nozeros_compensation is %d, intervalSize is %d\n",num_nozeros_compensation, interval_size2 );
	if (interval_size2>2*32)
	{
		//512*512
		size_t sizeKernel0=(block_size*thread_size/WARP_SIZE)*sizeof(unsigned int);
		size_t sizeKernel1=(block_size*thread_size/WARP_SIZE)*sizeof(double);		
		int *temp_rows1;
		double *temp_vals1;
		int *temp_rows2;
		double *temp_vals2;
		hipMalloc((void**)&temp_rows1, sizeKernel0);
		hipMalloc((void**)&temp_vals1, sizeKernel1);
		hipMalloc((void**)&temp_rows2, block_size2*sizeof(unsigned int));
		hipMalloc((void**)&temp_vals2, block_size2*sizeof(double));
		COO_level1<<<block_size,thread_size>>>(num_nozeros_compensation,interval_size2, 
					I, J, V, x_d, y_d, temp_rows1, temp_vals1, bias0, bias1);
		COO_level2<<<block_size2,thread_size2>>>(temp_rows1,temp_vals1,temp_rows2,temp_vals2,y_d, bias1);
		COO_level3<<<1,1>>>(block_size2,temp_rows2,temp_vals2,y_d, bias1);
		//COO_level2_serial<<<1,1>>>(temp_rows1,temp_vals1,y_d, bias1);
	}
	else if (interval_size2>1)
	//if (interval_size2>32)
	{
		//16*512
		//printf("situation 2 happened!\n");
		size_t sizeKernel2=(block_size2*thread_size2/WARP_SIZE)*sizeof(unsigned int);
		size_t sizeKernel3=(block_size2*thread_size2/WARP_SIZE)*sizeof(double);
		int *temp_rows3;
		double *temp_vals3;
		hipMalloc((void**)&temp_rows3, sizeKernel2);
		hipMalloc((void**)&temp_vals3, sizeKernel3);
		COO_level1<<<block_size2,thread_size2>>>(num_nozeros_compensation, interval_size2, 
				I, J, V, x_d, y_d, temp_rows3, temp_vals3, bias0, bias1);
		//512 calculation excuted serially
		COO_level2_serial2<<<1,1>>>(temp_rows3,temp_vals3,y_d, bias1);		
	}
	/*else if (interval_size2>4)
	{
		//16*32
		unsigned int iterval_size3=ceil((double) num_nozeros_compensation/(512*2/32));
		unsigned int *temp_rows4;
		double *temp_vals4;
		hipMalloc((void**)&temp_rows4, 32*sizeof(unsigned int));
		hipMalloc((void**)&temp_vals4, 32*sizeof(double));		
		COO_level1<<<2,512>>>(num_nozeros_compensation, iterval_size3, I,J,V,x_d,y_d,temp_rows4,temp_vals4,bias0,bias1);
		//16 calculation excuted serially
		COO_level2_serial3<<<1,1>>>(32, temp_rows4,temp_vals4,y_d, bias1);		
	}*/
	else
	{
		//less than 512, all calculation excuted serially
		//printf("situation 3 happen\n");
		COO_level1_serial<<<1,1>>>(num_nozeros_compensation, I,J,V,x_d,y_d, bias0,bias1);
	}
	
}

/*void matrix_vectorHYP(const unsigned int num_rows, const unsigned int max, const unsigned int *J, const double *V, unsigned int NumCOO, unsigned int *I_COO, unsigned int *J_COO, double *V_COO, )
{
	matrix_vectorELL();
	matrix_vectorCOO();
}*/
