#include "hip/hip_runtime.h"
#include "kernel.h"

#define FULL_MASK 0xffffffff
#define BASE 262144 //1024*1024

#define block_size 512	
#define thread_size 512
#define block_size2 16
#define thread_size2 512
#define WARP_SIZE 32

static __inline__ __device__ double fetch_double(uint2 p){
	    return __hiloint2double(p.y, p.x);
}

texture<uint2, 1> texInput;
texture<float, 1> texInputFloat;

static bool texInited = false;

/*kernel function for initialize*/
__global__ void kernelInitialize(const uint32_t num, double *x)
{
	uint32_t idx=blockDim.x * blockIdx.x+ threadIdx.x;
	
	for (uint32_t n=idx;n<num;n+=BASE) x[n]=0;
}

__global__ void kernelInitializeAll(const uint32_t num, double *pk, double *bp, double *x, double *zk, const double *vector_in)
{
	uint32_t idx=blockDim.x * blockIdx.x+ threadIdx.x;
	double temp;
	for (uint32_t n=idx;n<num;n+=BASE) 
	{
		temp=zk[n];
		pk[n]=temp;
		bp[n]=0;
		x[n]=0;
	}
}

__global__ void kernelInitializeR(const uint32_t num,double *rk, const double *vector_in)
{
	uint32_t idx=blockDim.x * blockIdx.x+ threadIdx.x;
	double temp;
	for (uint32_t n=idx;n<num;n+=BASE) 
	{
		temp=vector_in[n];
		rk[n]=temp;
	}
}

//for ELL format matrix, output y=data*x
__global__ void ELL_kernel(const uint32_t num_rows, const uint32_t cal_rows, const uint32_t num_cols_per_row,
			const uint32_t *indices, const double *data, const double * x, double * y) 
{
	uint32_t row= blockDim.x*blockIdx.x+threadIdx.x;
	if (row<cal_rows){
		double dot =0;
		
		for (uint32_t n=0; n< num_cols_per_row; n++){
			uint32_t col=indices[num_rows * n + row];
			double val=data[num_rows*n+row];

			if (val != 0)
				dot += val* x[col];
		}
			y[row]=dot;
	}
}
/*
__global__ void ELL_kernel_float(const uint32_t num_rows, const uint32_t cal_rows, const uint32_t num_cols_per_row,
			const uint32_t *indices, const float* data, const double * x, double * y) 
			
{
	uint32_t row= blockDim.x*blockIdx.x+threadIdx.x;
	if (row<cal_rows){
		double dot =0;
		for (uint32_t n=0; n< num_cols_per_row; n++){
			uint32_t col=indices[num_rows * n + row];
			double val=data[num_rows*n+row];

			if (val != 0)
				dot += val* x[col];
		}
		y[row]+=dot;
	}
}*/
__global__ void ELL_kernel_block(const uint32_t num_rows,
			const uint32_t* num_cols_per_row_vec, 
			const uint32_t* block_data_bias_vec,  
			const uint32_t *indices, 
			const double *data, const double * x, double * y,
			bool tex = false) 
{
	
	uint32_t block_idx = blockIdx.x; 
	uint32_t thread_idx = threadIdx.x; 
	uint32_t num_cols_per_row;  
	uint32_t block_data_bias;
	uint32_t data_idx;
	uint32_t col;	
	double val;
	uint32_t row= blockDim.x*blockIdx.x+threadIdx.x;
	if(row < num_rows){
		num_cols_per_row = num_cols_per_row_vec[block_idx];//cache will work when every threads read same global address
		block_data_bias = block_data_bias_vec[block_idx];
		double dot =0;
		for (uint32_t n=0; n< num_cols_per_row; n++){
			data_idx = block_data_bias + ELL_threadSize*n + thread_idx;
			col=indices[data_idx];
			val=data[data_idx];

			if (val != 0){
				if(tex == false)
					dot += val* x[col];
				else
					dot += val*fetch_double(tex1Dfetch(texInput, col));
			}
		}
		y[row]+=dot;
	}
}

__global__ void ELL_kernel_rodr(const uint32_t* num_cols_per_row_vec,
		const uint32_t* block_data_bias_vec,  
		const uint32_t *indices, const double *data, const double * x,
		double * y,
		const uint32_t* part_boundary,
		const bool tex)
{
	uint32_t part_idx = blockIdx.x; 
	uint32_t x_idx = threadIdx.x;
	uint32_t vec_start = part_boundary[blockIdx.x];
	uint32_t vec_end = part_boundary[blockIdx.x + 1];
	uint32_t row = 0;
	double val, dot;
	uint32_t block_idx, data_idx, col;
	uint32_t block_rowSize, block_data_bias, num_cols_per_row;
	uint32_t block_base = part_idx * block_per_part;
		
	uint32_t endBlockRow;
	if(vec_end >= vec_start+block_per_part*ELL_threadSize)
		endBlockRow = ELL_threadSize; 
	else 
		endBlockRow = vec_end - (vec_start+(block_per_part - 1)*ELL_threadSize);

	for(uint32_t i = 0; i < block_per_part; ++i){//the thread is step with stride ELL_threadSize
		dot =0;
		block_rowSize = ELL_threadSize;
		block_idx = block_base + i;
		if(i == (block_per_part -1)){
			block_rowSize = endBlockRow;
		}
		block_data_bias = block_data_bias_vec[block_idx];
		num_cols_per_row = num_cols_per_row_vec[block_idx];//cache will work if it is shared by all threads
		row = i*ELL_threadSize + vec_start + x_idx;
		if(row < vec_end){
			for(uint32_t n=0; n< num_cols_per_row; ++n){
				data_idx = block_data_bias + block_rowSize*n + x_idx;//however the data storage is stride with block_rowSize
				col=indices[data_idx];
				val=data[data_idx];
				//if(tex == false)
					dot += val* x[col];
				//else
				//	dot += fetch_double(tex1Dfetch(texInput, col));

			}
			y[row] = dot;
		}
		block_idx += 1;
	}		
}

__global__ void ELL_kernel_rodr_test(const uint32_t* num_cols_per_row_vec,
		const uint32_t* block_data_bias_vec,  
		const uint32_t *indices, const double *data, const double * x,
		double * y,
		const uint32_t* part_boundary,
		const uint32_t testPoint)
{
	uint32_t part_idx = blockIdx.x; 
	uint32_t x_idx = threadIdx.x;
	uint32_t vec_start = part_boundary[blockIdx.x];
	uint32_t vec_end = part_boundary[blockIdx.x + 1];
	uint32_t row = 0;
	double val, dot;
	uint32_t block_idx, data_idx, col;
	uint32_t block_rowSize, block_data_bias, num_cols_per_row;
	uint32_t block_base = part_idx * block_per_part;
		
	uint32_t endBlockRow;
	if(vec_end >= vec_start+block_per_part*ELL_threadSize)
		endBlockRow = ELL_threadSize; 
	else 
		endBlockRow = vec_end - (vec_start+(block_per_part - 1)*ELL_threadSize);


	for(uint32_t i = 0; i < block_per_part; ++i){//the thread is step with stride ELL_threadSize
		dot =0;
		block_rowSize = ELL_threadSize;
		block_idx = block_base + i;
		if(i == (block_per_part -1)){
			block_rowSize = endBlockRow;
		}
		block_data_bias = block_data_bias_vec[block_idx];
		num_cols_per_row = num_cols_per_row_vec[block_idx];
		row = i*ELL_threadSize + vec_start + x_idx;
		if(row < vec_end){
			for(uint32_t n=0; n< num_cols_per_row; ++n){
				data_idx = block_data_bias + block_rowSize*n + x_idx;//however the data storage is stride with block_rowSize
				col=indices[data_idx];
				val=data[data_idx];
				if(val != 0){
					if(row == testPoint)
						dot = dot + val* x[col];
					else
						dot += val* x[col];
				}
			}
			if(row == testPoint)
				y[row] = dot + 1 - 0.999;
			else
				y[row] = dot;
		}
		block_idx += 1;
	}		
}

__global__ void ELL_cached_kernel_rodr(const uint32_t* num_cols_per_row_vec,
		const uint32_t* block_data_bias_vec,  
		const uint32_t *indices, const double *data, const double * x,
		double * y,
		const uint32_t* part_boundary,
		const bool tex)
{
	uint32_t part_idx = blockIdx.x; 
	uint32_t x_idx = threadIdx.x;
	__shared__ volatile double cached_vec[vector_cache_size];  
	uint32_t vec_start = part_boundary[blockIdx.x];
	uint32_t vec_end = part_boundary[blockIdx.x + 1];
	uint32_t row = 0;

	for (uint32_t i = x_idx; i < vector_cache_size; i += ELL_threadSize){
		if(i < vec_end) cached_vec[i] = x[i + vec_start];
		else cached_vec[i] = 0;
	}
	double val, fetched, dot;
	uint32_t block_idx, data_idx, col;
	uint32_t block_rowSize, block_data_bias, num_cols_per_row;
	uint32_t block_base = part_idx * block_per_part;
		
	uint32_t endBlockRow;
	if(vec_end >= vec_start+block_per_part*ELL_threadSize)
		endBlockRow = ELL_threadSize; 
	else 
		endBlockRow = vec_end - (vec_start+(block_per_part - 1)*ELL_threadSize);

	for(uint32_t i = 0; i < block_per_part; ++i){//the thread is step with stride ELL_threadSize
		dot =0;
		block_rowSize = ELL_threadSize;
		block_idx = block_base + i;
		if(i == block_per_part -1){
			block_rowSize = endBlockRow;
		}
		block_data_bias = block_data_bias_vec[block_idx];
		num_cols_per_row = num_cols_per_row_vec[block_idx];
		row = i*ELL_threadSize + vec_start + x_idx;
		if(row < vec_end){
			for(uint32_t n=0; n< num_cols_per_row; ++n){
				data_idx = block_data_bias + block_rowSize*n + x_idx;//however the data storage is stride with block_rowSize
				col=indices[data_idx];
				val=data[data_idx];
				if(val != 0){
					if(col > vec_start && col < vec_start + vector_cache_size)
						fetched = cached_vec[col - vec_start];
					else{
						if(tex == false)
							fetched = x[col];
						else
							fetched = fetch_double(tex1Dfetch(texInput, col));
					}
					dot += val*fetched;
				}
			}
			y[row] = dot;
		}
		block_idx += 1;
	}		
}

__global__ void COO_shared(const uint32_t num_nozeros, const uint32_t interval_size,
				const uint32_t *I, const uint32_t *J, const double *V,
				const double *x, double *y)
{
	__shared__ volatile int rows[48*thread_size/WARP_SIZE];  //why using 48? because we need 16 additional junk elements
	__shared__ volatile double vals[thread_size];

	uint32_t thread_id = blockDim.x*blockIdx.x + threadIdx.x;
	uint32_t thread_lane= threadIdx.x & (WARP_SIZE-1); //great idea! think about it
	uint32_t warp_id = thread_id / WARP_SIZE;

	uint32_t interval_begin=warp_id*interval_size;
	uint32_t interval_end =min(interval_begin+interval_size, num_nozeros);
	/*how about the interval is not the multiple of warp_size?*/
	//uint32_t iteration_end=((interval_end)/WARP_SIZE)*WARP_SIZE;

	uint32_t idx=16*(threadIdx.x/32+1) + threadIdx.x;//every warp has 16 "junk" rows elements

	rows[idx-16]=-1;
	
	uint32_t n;
	n=interval_begin+thread_lane;
	while (n< interval_end)
	{
		uint32_t row =I[n];
		//double val=V[n]*fetch_x(J[n], x);
		double val=V[n]*x[J[n]];

		
		rows[idx] =row;
		vals[threadIdx.x] =val;

        if(row == rows[idx -  1]) { vals[threadIdx.x] = val = val + vals[threadIdx.x -  1]; }
        if(row == rows[idx -  2]) { vals[threadIdx.x] = val = val + vals[threadIdx.x -  2]; }
        if(row == rows[idx -  4]) { vals[threadIdx.x] = val = val + vals[threadIdx.x -  4]; }
        if(row == rows[idx -  8]) { vals[threadIdx.x] = val = val + vals[threadIdx.x -  8]; }
        if(row == rows[idx - 16]) { vals[threadIdx.x] = val = val + vals[threadIdx.x - 16]; }

		if(thread_lane == 31 || n == interval_end -1){
			//if(row == testPoint){
			//	y[row] += val;
			//}else
				atomicAdd(&y[row],val);  
		}else{
			if(row != rows[idx + 1]){
			//	if(row == testPoint ){
			//		y[row] += val;
			//	} else
					y[row] += val;
					//atomicAdd(&y[row],val);  
				
			}
		}
        
		n+=WARP_SIZE;
	}
	

}



__global__ void COO_atomic(const uint32_t num_nozeros, const uint32_t interval_size, 
				const uint32_t *I, const uint32_t *J, const double *V, 
				const double *x, double *y, bool tex, uint32_t testPoint) 
{
	
	uint32_t thread_id = blockDim.x*blockIdx.x + threadIdx.x;
	uint32_t thread_lane= threadIdx.x & (WARP_SIZE-1); //great idea! think about it
	uint32_t warp_id = thread_id / WARP_SIZE;
	
	uint32_t interval_begin=warp_id*interval_size;
	uint32_t interval_end =min(interval_begin+interval_size, num_nozeros);
	/*how about the interval is not the multiple of warp_size?*/
	//uint32_t iteration_end=((interval_end)/WARP_SIZE)*WARP_SIZE;
	int row;
	double val;
	uint32_t row_tmp;
	double val_tmp;
	
	uint32_t n;
	n=interval_begin+thread_lane;
	while (n< interval_end)
	{
		row = I[n];
		val = V[n]*x[J[n]];
		
		//double val=V[n]*x[J[n]];
		val_tmp = __shfl_up_sync(FULL_MASK, val, 1);
		row_tmp = __shfl_up_sync(FULL_MASK, row, 1);
		if(thread_lane > 0 && row == row_tmp) { 
			val += val_tmp; 
		} 
		val_tmp = __shfl_up_sync(FULL_MASK, val, 2);
		row_tmp = __shfl_up_sync(FULL_MASK, row, 2);
		if(thread_lane > 1 && row == row_tmp) { 
			val += val_tmp; 
		}
		val_tmp = __shfl_up_sync(FULL_MASK, val, 4);
		row_tmp = __shfl_up_sync(FULL_MASK, row, 4);
		if(thread_lane > 3 && row == row_tmp) { 
			val += val_tmp; 
		}
		val_tmp = __shfl_up_sync(FULL_MASK, val, 8);
		row_tmp = __shfl_up_sync(FULL_MASK, row, 8);
		if(thread_lane > 7 && row == row_tmp) { 
			val += val_tmp; 
		}
		val_tmp = __shfl_up_sync(FULL_MASK, val, 16);
		row_tmp = __shfl_up_sync(FULL_MASK, row, 16);
		if(thread_lane > 15 && row == row_tmp) { 
			val += val_tmp; 
		}
		row_tmp = __shfl_down_sync(FULL_MASK, row, 1);
		if(thread_lane == 31 || n == interval_end -1){
			//if(row == testPoint){
			//	y[row] += val;
			//}else
				atomicAdd(&y[row],val);  
		}else{
			if(row != row_tmp){
			//	if(row == testPoint ){
			//		y[row] += val;
			//	} else
					y[row] += val;
					//atomicAdd(&y[row],val);  
				
			}
		}	
		n+=WARP_SIZE;
	}
	
	
}


//y=x+gamak*y
__global__ void kernelMyxpy(const uint32_t dimension, double gamak, const double *x, double *y)
{
	uint32_t idx=blockDim.x*blockIdx.x+threadIdx.x;
	uint32_t n=idx;
	while(n<dimension){
		y[n]=x[n]+gamak*y[n];
		n=n+BASE;
	}
}
extern "C"
void initialize_all(const uint32_t dimension, double *pk_d, double *bp_d, double *x, double *zk, const double *vector_in_d)
{
	kernelInitializeAll<<<block_size,thread_size>>>(dimension, pk_d, bp_d, x, zk, vector_in_d);
}

void initialize_bp(uint32_t num, double *x)
{
	kernelInitialize<<<block_size,thread_size>>>(num,x);
}

void initialize_r(uint32_t num, double *rk, double *vector_in)
{
	kernelInitializeR<<<block_size,thread_size>>>(num,rk,vector_in);
}
void myxpy(const uint32_t dimension, double gamak, const double *x, double *y)
{
	kernelMyxpy<<<block_size,thread_size>>>(dimension,gamak,x,y);
}

void initialDeviceArray(uint32_t num, double *x)
{
	kernelInitialize<<<512,512>>>(num,x);
}


void matrix_vectorELL(const uint32_t num_rows, const uint32_t cal_rows, 
			const uint32_t num_cols_per_row,  const uint32_t *J,
 			const double *V, const double *x, double *y,
			const bool RODR, const uint32_t rodr_blocks, const uint32_t* part_boundary_d)
{
	uint32_t ELL_blocks = ceil((double) num_rows/ELL_threadSize);
	//printf("ELL_blocks is %d\n", ELL_blocks);
	//bind_x(x);
	ELL_kernel<<<ELL_blocks, ELL_threadSize>>>(num_rows, cal_rows, num_cols_per_row, J, V, x,y);
	//unbind_x(x);
	
}

void matrix_vectorELL_block(const uint32_t num_rows, const uint32_t testPoint, 
			const uint32_t* num_cols_per_row_vec, 
			const uint32_t* block_data_bias_vec,    
			const uint32_t *J,
 			const double *V, const double *x, double *y,
			const bool CACHE, const uint32_t rodr_blocks, const uint32_t* part_boundary_d,
			const bool tex=false)
{
	uint32_t ELL_blocks = ceil((double) num_rows/ELL_threadSize);
	//printf("ELL_blocks is %d\n", ELL_blocks);
	//bind_x(x);
	
	if(rodr_blocks > 0){
		if(CACHE){	
			ELL_cached_kernel_rodr<<<rodr_blocks, ELL_threadSize>>>(num_cols_per_row_vec, 
					block_data_bias_vec,
					J, V, x, y, part_boundary_d, tex);
			gpuErrchk( hipPeekAtLastError() );
		} else {
			if(testPoint > 0){
				ELL_kernel_rodr_test<<<rodr_blocks, ELL_threadSize>>>(num_cols_per_row_vec, 
						block_data_bias_vec,
						J, V, x, y, part_boundary_d, testPoint);
			} else {
				ELL_kernel_rodr<<<rodr_blocks, ELL_threadSize>>>(num_cols_per_row_vec, 
						block_data_bias_vec,
						J, V, x, y, part_boundary_d, tex);
			}
			
		}

	}else{
		ELL_kernel_block<<<ELL_blocks, ELL_threadSize>>>(num_rows, num_cols_per_row_vec, 
			block_data_bias_vec, J, V, x,y);
	}

		
	//unbind_x(x);
	
}

void matrix_vectorCOO(const uint32_t num_nozeros_compensation, uint32_t *I, uint32_t *J, double *V, double *x_d, double *y_d, uint32_t testPoint, bool tex=false)
{
	uint32_t interval_size2;
	interval_size2=ceil(((double) num_nozeros_compensation)/(512*512/WARP_SIZE));//for data with 2 million elements, we have interval size 200	
	//COO_atomic<<<512, 512>>>(num_nozeros_compensation, interval_size2, I, J, V, x_d, y_d, tex, testPoint);
	COO_shared<<<512, 512>>>(num_nozeros_compensation, interval_size2, I, J, V, x_d, y_d);

}

void matrix_vectorHYB(matrixHYB_S_d* inputMatrix, double* vector_in_d,
		double* vector_out_d, cb_s cb, const uint32_t testPoint,
		const uint32_t part_size, const uint32_t* part_boundary_d, 
		const bool tex=false)
{
	uint32_t dimension = inputMatrix->dimension;
	uint32_t ELL_width = inputMatrix->ELL_width;
	uint32_t totalNumCOO = inputMatrix->totalNumCOO;
	uint32_t* col_d = inputMatrix->col_d;
	uint32_t* I_COO_d = inputMatrix->I_COO_d;
	uint32_t* J_COO_d = inputMatrix->J_COO_d;
	double* V_d = inputMatrix->V_d;
	double* V_COO_d = inputMatrix->V_COO_d;
	uint32_t* ELL_block_bias_vec_d = inputMatrix->ELL_block_bias_vec_d;
	uint32_t* ELL_block_cols_vec_d = inputMatrix->ELL_block_cols_vec_d;
	size_t offset = 0;
	if(tex==true){
		if(texInited == false){
			texInput.addressMode[0] = hipAddressModeBorder;
			texInput.addressMode[1] = hipAddressModeBorder;
			texInput.filterMode = hipFilterModePoint;
			texInput.normalized = false;
			texInited = true;
		}
		hipBindTexture(&offset, texInput, vector_in_d, sizeof(double)*dimension);	
	}
	if(!cb.BLOCK){
		matrix_vectorELL(dimension, dimension, ELL_width, col_d,V_d,
				vector_in_d, vector_out_d, false, 0, NULL);
	} else {
		if(cb.RODR){
			matrix_vectorELL_block(dimension, 0, ELL_block_cols_vec_d, 
					ELL_block_bias_vec_d,
					col_d,V_d, vector_in_d, vector_out_d,
					cb.CACHE, part_size, part_boundary_d, tex);
		}
		else{
			matrix_vectorELL_block(dimension, 0, ELL_block_cols_vec_d, 
					ELL_block_bias_vec_d,
					col_d, V_d, 
					vector_in_d, vector_out_d,
					false, 0, NULL, tex);
		}
	}

	if (totalNumCOO > 0) matrix_vectorCOO(totalNumCOO, I_COO_d, J_COO_d, V_COO_d, 
			vector_in_d, vector_out_d, testPoint, tex);

	if(tex==true)
		hipUnbindTexture(texInput);
}

