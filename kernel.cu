#include "hip/hip_runtime.h"
#include "kernel.h"

#define FULL_MASK 0xffffffff
#define BASE 262144 //1024*1024

#define block_size 512	
#define thread_size 512
#define block_size2 16
#define thread_size2 512
#define WARP_SIZE 32


texture<double, 1, hipReadModeElementType> texInput;

texInput.addressMode[0] = hipAddressModeBorder;
texInput.addressMode[1] = hipAddressModeBorder;
texInput.filterMode = hipFilterModePoint;
texInput.normalized = false;


/*the device function for level 2 reduce*/
__device__ void segreduce_block(const int * idx, double * val)
{
    double left = 0;
    if( threadIdx.x >=   1 && idx[threadIdx.x] == idx[threadIdx.x -   1] ) { left = val[threadIdx.x -   1]; } __syncthreads(); val[threadIdx.x] += left; left = 0; __syncthreads();  
    if( threadIdx.x >=   2 && idx[threadIdx.x] == idx[threadIdx.x -   2] ) { left = val[threadIdx.x -   2]; } __syncthreads(); val[threadIdx.x] += left; left = 0; __syncthreads();
    if( threadIdx.x >=   4 && idx[threadIdx.x] == idx[threadIdx.x -   4] ) { left = val[threadIdx.x -   4]; } __syncthreads(); val[threadIdx.x] += left; left = 0; __syncthreads();
    if( threadIdx.x >=   8 && idx[threadIdx.x] == idx[threadIdx.x -   8] ) { left = val[threadIdx.x -   8]; } __syncthreads(); val[threadIdx.x] += left; left = 0; __syncthreads();
    if( threadIdx.x >=  16 && idx[threadIdx.x] == idx[threadIdx.x -  16] ) { left = val[threadIdx.x -  16]; } __syncthreads(); val[threadIdx.x] += left; left = 0; __syncthreads();
    if( threadIdx.x >=  32 && idx[threadIdx.x] == idx[threadIdx.x -  32] ) { left = val[threadIdx.x -  32]; } __syncthreads(); val[threadIdx.x] += left; left = 0; __syncthreads();  
    if( threadIdx.x >=  64 && idx[threadIdx.x] == idx[threadIdx.x -  64] ) { left = val[threadIdx.x -  64]; } __syncthreads(); val[threadIdx.x] += left; left = 0; __syncthreads();
    if( threadIdx.x >= 128 && idx[threadIdx.x] == idx[threadIdx.x - 128] ) { left = val[threadIdx.x - 128]; } __syncthreads(); val[threadIdx.x] += left; left = 0; __syncthreads();
    if( threadIdx.x >= 256 && idx[threadIdx.x] == idx[threadIdx.x - 256] ) { left = val[threadIdx.x - 256]; } __syncthreads(); val[threadIdx.x] += left; left = 0; __syncthreads();
	if( threadIdx.x >= 512 && idx[threadIdx.x] == idx[threadIdx.x - 512] ) { left = val[threadIdx.x - 512]; } __syncthreads(); val[threadIdx.x] += left; left = 0; __syncthreads();
}

__device__ 
double get_val(const uint32_t idx, const uint32_t scope1, const uint32_t scope2,  const double *vec, volatile double* cached_vec){
	if(idx > scope1 && idx < scope2)
		return cached_vec[idx - scope1];
	else
		return vec[idx];
}

/*kernel function for initialize*/
__global__ void kernelInitialize(const uint32_t num, double *x)
{
	uint32_t idx=blockDim.x * blockIdx.x+ threadIdx.x;
	
	for (uint32_t n=idx;n<num;n+=BASE) x[n]=0;
}

__global__ void kernelInitializeAll(const uint32_t num, double *pk, double *bp, double *x, double *zk, const double *vector_in)
{
	uint32_t idx=blockDim.x * blockIdx.x+ threadIdx.x;
	double temp;
	for (uint32_t n=idx;n<num;n+=BASE) 
	{
		temp=zk[n];
		pk[n]=temp;
		bp[n]=0;
		x[n]=0;
	}
}

__global__ void kernelInitializeR(const uint32_t num,double *rk, const double *vector_in)
{
	uint32_t idx=blockDim.x * blockIdx.x+ threadIdx.x;
	double temp;
	for (uint32_t n=idx;n<num;n+=BASE) 
	{
		temp=vector_in[n];
		rk[n]=temp;
	}
}

//for ELL format matrix, output y=data*x
__global__ void ELL_kernel(const uint32_t num_rows, const uint32_t cal_rows, const uint32_t num_cols_per_row,
			const uint32_t *indices, const double *data, const double * x, double * y) 
{
	uint32_t row= blockDim.x*blockIdx.x+threadIdx.x;
	if (row<cal_rows){
		double dot =0;
		
		for (uint32_t n=0; n< num_cols_per_row; n++){
			uint32_t col=indices[num_rows * n + row];
			double val=data[num_rows*n+row];

			if (val != 0)
				dot += val* x[col];
		}
			y[row]=dot;
	}
}
/*
__global__ void ELL_kernel_float(const uint32_t num_rows, const uint32_t cal_rows, const uint32_t num_cols_per_row,
			const uint32_t *indices, const float* data, const double * x, double * y) 
			
{
	uint32_t row= blockDim.x*blockIdx.x+threadIdx.x;
	if (row<cal_rows){
		double dot =0;
		for (uint32_t n=0; n< num_cols_per_row; n++){
			uint32_t col=indices[num_rows * n + row];
			double val=data[num_rows*n+row];

			if (val != 0)
				dot += val* x[col];
		}
		y[row]+=dot;
	}
}*/
__global__ void ELL_kernel_block(const uint32_t num_rows,
			const uint32_t* num_cols_per_row_vec, 
			const uint32_t* block_data_bias_vec,  
			const uint32_t *indices, 
			const double *data, const double * x, double * y) 
{
	
	uint32_t block_idx = blockIdx.x; 
	uint32_t thread_idx = threadIdx.x; 
	uint32_t num_cols_per_row;  
	uint32_t block_data_bias;
	uint32_t data_idx;
	uint32_t col;	
	double val;
	uint32_t row= blockDim.x*blockIdx.x+threadIdx.x;
	if(row < num_rows){
		num_cols_per_row = num_cols_per_row_vec[block_idx];//cache will work when every threads read same global address
		block_data_bias = block_data_bias_vec[block_idx];
		double dot =0;
		for (uint32_t n=0; n< num_cols_per_row; n++){
			data_idx = block_data_bias + ELL_threadSize*n + thread_idx;
			col=indices[data_idx];
			val=data[data_idx];

			if (val != 0)
				dot += val* x[col];
		}
		y[row]+=dot;
	}
}

__global__ void ELL_kernel_rodr(const uint32_t* num_cols_per_row_vec,
		const uint32_t* block_data_bias_vec,  
		const uint32_t *indices, const double *data, const double * x,
		double * y,
		const uint32_t* part_boundary,
		const bool tex)
{
	uint32_t part_idx = blockIdx.x; 
	uint32_t x_idx = threadIdx.x;
	uint32_t vec_start = part_boundary[blockIdx.x];
	uint32_t vec_end = part_boundary[blockIdx.x + 1];
	uint32_t row = 0;
	double val, dot;
	uint32_t block_idx, data_idx, col;
	uint32_t block_rowSize, block_data_bias, num_cols_per_row;
	uint32_t block_base = part_idx * block_per_part;
		
	uint32_t endBlockRow;
	if(vec_end >= vec_start+block_per_part*ELL_threadSize)
		endBlockRow = ELL_threadSize; 
	else 
		endBlockRow = vec_end - (vec_start+(block_per_part - 1)*ELL_threadSize);

	for(uint32_t i = 0; i < block_per_part; ++i){//the thread is step with stride ELL_threadSize
		dot =0;
		block_rowSize = ELL_threadSize;
		block_idx = block_base + i;
		if(i == (block_per_part -1)){
			block_rowSize = endBlockRow;
		}
		block_data_bias = block_data_bias_vec[block_idx];
		num_cols_per_row = num_cols_per_row_vec[block_idx];//cache will work if it is shared by all threads
		row = i*ELL_threadSize + vec_start + x_idx;
		if(row < vec_end){
			for(uint32_t n=0; n< num_cols_per_row; ++n){
				data_idx = block_data_bias + block_rowSize*n + x_idx;//however the data storage is stride with block_rowSize
				col=indices[data_idx];
				val=data[data_idx];
				if(tex == false)
					dot += val* x[col];
				else
					dot += tex1Dfetch(texInput, col);

			}
			y[row] = dot;
		}
		block_idx += 1;
	}		
}

__global__ void ELL_kernel_rodr_test(const uint32_t* num_cols_per_row_vec,
		const uint32_t* block_data_bias_vec,  
		const uint32_t *indices, const double *data, const double * x,
		double * y,
		const uint32_t* part_boundary,
		const uint32_t testPoint)
{
	uint32_t part_idx = blockIdx.x; 
	uint32_t x_idx = threadIdx.x;
	uint32_t vec_start = part_boundary[blockIdx.x];
	uint32_t vec_end = part_boundary[blockIdx.x + 1];
	uint32_t row = 0;
	double val, dot;
	uint32_t block_idx, data_idx, col;
	uint32_t block_rowSize, block_data_bias, num_cols_per_row;
	uint32_t block_base = part_idx * block_per_part;
		
	uint32_t endBlockRow;
	if(vec_end >= vec_start+block_per_part*ELL_threadSize)
		endBlockRow = ELL_threadSize; 
	else 
		endBlockRow = vec_end - (vec_start+(block_per_part - 1)*ELL_threadSize);


	for(uint32_t i = 0; i < block_per_part; ++i){//the thread is step with stride ELL_threadSize
		dot =0;
		block_rowSize = ELL_threadSize;
		block_idx = block_base + i;
		if(i == (block_per_part -1)){
			block_rowSize = endBlockRow;
		}
		block_data_bias = block_data_bias_vec[block_idx];
		num_cols_per_row = num_cols_per_row_vec[block_idx];
		row = i*ELL_threadSize + vec_start + x_idx;
		if(row < vec_end){
			for(uint32_t n=0; n< num_cols_per_row; ++n){
				data_idx = block_data_bias + block_rowSize*n + x_idx;//however the data storage is stride with block_rowSize
				col=indices[data_idx];
				val=data[data_idx];
				if(val != 0){
					if(row == testPoint)
						dot = dot + val* x[col];
					else
						dot += val* x[col];
				}
			}
			if(row == testPoint)
				y[row] = dot + 1 - 0.999;
			else
				y[row] = dot;
		}
		block_idx += 1;
	}		
}

__global__ void ELL_cached_kernel_rodr(const uint32_t* num_cols_per_row_vec,
		const uint32_t* block_data_bias_vec,  
		const uint32_t *indices, const double *data, const double * x,
		double * y,
		const uint32_t* part_boundary,
		const bool tex)
{
	uint32_t part_idx = blockIdx.x; 
	uint32_t x_idx = threadIdx.x;
	__shared__ volatile double cached_vec[vector_cache_size];  
	uint32_t vec_start = part_boundary[blockIdx.x];
	uint32_t vec_end = part_boundary[blockIdx.x + 1];
	uint32_t row = 0;

	for (uint32_t i = x_idx; i < vector_cache_size; i += ELL_threadSize){
		if(i < vec_end) cached_vec[i] = x[i + vec_start];
		else cached_vec[i] = 0;
	}
	double val, fetched, dot;
	uint32_t block_idx, data_idx, col;
	uint32_t block_rowSize, block_data_bias, num_cols_per_row;
	uint32_t block_base = part_idx * block_per_part;
		
	uint32_t endBlockRow;
	if(vec_end >= vec_start+block_per_part*ELL_threadSize)
		endBlockRow = ELL_threadSize; 
	else 
		endBlockRow = vec_end - (vec_start+(block_per_part - 1)*ELL_threadSize);

	for(uint32_t i = 0; i < block_per_part; ++i){//the thread is step with stride ELL_threadSize
		dot =0;
		block_rowSize = ELL_threadSize;
		block_idx = block_base + i;
		if(i == block_per_part -1){
			block_rowSize = endBlockRow;
		}
		block_data_bias = block_data_bias_vec[block_idx];
		num_cols_per_row = num_cols_per_row_vec[block_idx];
		row = i*ELL_threadSize + vec_start + x_idx;
		if(row < vec_end){
			for(uint32_t n=0; n< num_cols_per_row; ++n){
				data_idx = block_data_bias + block_rowSize*n + x_idx;//however the data storage is stride with block_rowSize
				col=indices[data_idx];
				val=data[data_idx];
				if(val != 0){
					if(col > vec_start && col < vec_start + vector_cache_size)
						fetched = cached_vec[col - vec_start];
					else{
						if(tex == false)
							fetched = x[col];
						else
							fetched = tex1Dfetch(x, col);
					}
					dot += val*fetched;
				}
			}
			y[row] = dot;
		}
		block_idx += 1;
	}		
}

/*
__global__ void ELL_cached_kernel_float(const uint32_t num_rows,  
				const uint32_t num_cols_per_row, 
				const uint32_t *indices, const float* data, const double * x,
				double * y,
				const uint32_t* part_boundary)
{
	uint32_t x_idx = blockDim.x*blockIdx.x+threadIdx.x;
	__shared__ volatile double cached_vec[vector_cache_size];  
	uint32_t vec_start = part_boundary[blockIdx.x];
	uint32_t vec_end = part_boundary[blockIdx.x + 1];
	double val, dot;
	uint32_t col;

	for (uint32_t i = x_idx; i < vector_cache_size; i += ELL_threadSize){
		cached_vec[i] = x[i + vec_start];
	}
	for(uint32_t row = x_idx; row < vec_end; row += ELL_threadSize){
		dot =0;
		for (uint32_t n=0; n< num_cols_per_row; n++){
			col=indices[num_rows*n + row];
			val=data[num_rows*n + row];
			if (val != 0)
				dot += val*get_val(col, vec_start, vec_start + vector_cache_size, x, cached_vec);
		}
		y[row] = dot;
	}		
}*/

__global__ void COO_atomic(const uint32_t num_nozeros, const uint32_t interval_size, 
				const uint32_t *I, const uint32_t *J, const double *V, 
				const double *x, double *y) 
{
	__shared__ volatile int rows[48*thread_size/WARP_SIZE];  //why using 48? because we need 16 additional junk elements
	__shared__ volatile double vals[thread_size];
	
	uint32_t thread_id = blockDim.x*blockIdx.x + threadIdx.x;
	uint32_t thread_lane= threadIdx.x & (WARP_SIZE-1); //great idea! think about it
	uint32_t warp_id = thread_id / WARP_SIZE;
	
	uint32_t interval_begin=warp_id*interval_size;
	uint32_t interval_end =min(interval_begin+interval_size, num_nozeros);
	/*how about the interval is not the multiple of warp_size?*/
	//uint32_t iteration_end=((interval_end)/WARP_SIZE)*WARP_SIZE;
	int row;
	double val;
	
	uint32_t n;
	n=interval_begin+thread_lane;
	while (n< interval_end)
	{
		uint32_t row =I[n];
		double val=V[n]*tex1Dfetch(texInput, I[n]);
		//double val=V[n]*x[J[n]];

        if(row == __shfl_up_sync(FULL_MASK, row, 1)) { val + __shfl_up_sync(FULL_MASK, val, 1); } 
        if(row == __shfl_up_sync(FULL_MASK, row, 2)) { val + __shfl_up_sync(FULL_MASK, val, 2); } 
        if(row == __shfl_up_sync(FULL_MASK, row, 4)) { val + __shfl_up_sync(FULL_MASK, val, 4); } 
        if(row == __shfl_up_sync(FULL_MASK, row, 8)) { val + __shfl_up_sync(FULL_MASK, val, 8); } 
        if(row == __shfl_up_sync(FULL_MASK, row, 16)) { val + __shfl_up_sync(FULL_MASK, val, 16); } 

        if(row != shfl_down_sync(FULL_MASK, row, 1 ) && n<interval_end-1)
           atomicAdd(&y[row],vals);  
		n+=WARP_SIZE;
	}
	
}

//for COO format matrix, output y=data*x
//the basic idea is come from
__global__ void COO_level1(const uint32_t num_nozeros, const uint32_t interval_size, 
				const uint32_t *I, const uint32_t *J, const double *V, 
				const double *x, double *y, int *temp_rows, 
				double *temp_vals)
{
	__shared__ volatile int rows[48*thread_size/WARP_SIZE];  //why using 48? because we need 16 additional junk elements
	__shared__ volatile double vals[thread_size];
	
	uint32_t thread_id = blockDim.x*blockIdx.x + threadIdx.x;
	uint32_t thread_lane= threadIdx.x & (WARP_SIZE-1); //great idea! think about it
	uint32_t warp_id = thread_id / WARP_SIZE;
	
	uint32_t interval_begin=warp_id*interval_size;
	uint32_t interval_end =min(interval_begin+interval_size, num_nozeros);
	/*how about the interval is not the multiple of warp_size?*/
	//uint32_t iteration_end=((interval_end)/WARP_SIZE)*WARP_SIZE;
	
	uint32_t idx=16*(threadIdx.x/32+1) + threadIdx.x;//every warp has 16 "junk" rows elements
	
	rows[idx-16]=-1;
	
	if(interval_begin >= interval_end)
	{
		temp_rows[warp_id] = -1;
		return;
	}	
	if (thread_lane ==31)
	{
		// initialize the cary in values
		rows[idx]=I[interval_begin];
		vals[threadIdx.x]=0;
	}
	uint32_t n;
	n=interval_begin+thread_lane;
	while (n< interval_end)
	{
		uint32_t row =I[n];
		//double val=V[n]*fetch_x(J[n], x);
		double val=V[n]*x[J[n]];
		
		if (thread_lane==0)
		{
			if (row==rows[idx+31])
				val+=vals[threadIdx.x+31]; //don't confused by the "plus" 31, because the former end is the new start
			else 
				y[rows[idx+31]] += vals[threadIdx.x+31];//try to fix the bug from orignial library functions
		}
		rows[idx] =row;
		vals[threadIdx.x] =val;
		
        if(row == rows[idx -  1]) { vals[threadIdx.x] = val = val + vals[threadIdx.x -  1]; } 
        if(row == rows[idx -  2]) { vals[threadIdx.x] = val = val + vals[threadIdx.x -  2]; }
        if(row == rows[idx -  4]) { vals[threadIdx.x] = val = val + vals[threadIdx.x -  4]; }
        if(row == rows[idx -  8]) { vals[threadIdx.x] = val = val + vals[threadIdx.x -  8]; }
        if(row == rows[idx - 16]) { vals[threadIdx.x] = val = val + vals[threadIdx.x - 16]; }

        if(thread_lane < 31 && row < rows[idx + 1] && n<interval_end-1)
            y[row] += vals[threadIdx.x];  
		n+=WARP_SIZE;
	}
	
	/*now we consider the reminder of interval_size/warp_size*/

    /*program at one warp is automatically sychronized*/
	if(n==(interval_end+WARP_SIZE-1))
    {
        // write the carry out values
        temp_rows[warp_id] = rows[idx];
        temp_vals[warp_id] = vals[threadIdx.x];
    }	
	
}



//y=x+gamak*y
__global__ void kernelMyxpy(const uint32_t dimension, double gamak, const double *x, double *y)
{
	uint32_t idx=blockDim.x*blockIdx.x+threadIdx.x;
	uint32_t n=idx;
	while(n<dimension){
		y[n]=x[n]+gamak*y[n];
		n=n+BASE;
	}
}
extern "C"
void initialize_all(const uint32_t dimension, double *pk_d, double *bp_d, double *x, double *zk, const double *vector_in_d)
{
	kernelInitializeAll<<<block_size,thread_size>>>(dimension, pk_d, bp_d, x, zk, vector_in_d);
}

void initialize_bp(uint32_t num, double *x)
{
	kernelInitialize<<<block_size,thread_size>>>(num,x);
}

void initialize_r(uint32_t num, double *rk, double *vector_in)
{
	kernelInitializeR<<<block_size,thread_size>>>(num,rk,vector_in);
}
void myxpy(const uint32_t dimension, double gamak, const double *x, double *y)
{
	kernelMyxpy<<<block_size,thread_size>>>(dimension,gamak,x,y);
}

void initialDeviceArray(uint32_t num, double *x)
{
	kernelInitialize<<<512,512>>>(num,x);
}


void matrix_vectorELL(const uint32_t num_rows, const uint32_t cal_rows, 
			const uint32_t num_cols_per_row,  const uint32_t *J,
 			const double *V, const double *x, double *y,
			const bool RODR, const uint32_t rodr_blocks, const uint32_t* part_boundary_d)
{
	uint32_t ELL_blocks = ceil((double) num_rows/ELL_threadSize);
	//printf("ELL_blocks is %d\n", ELL_blocks);
	//bind_x(x);
	ELL_kernel<<<ELL_blocks, ELL_threadSize>>>(num_rows, cal_rows, num_cols_per_row, J, V, x,y);
	//unbind_x(x);
	
}

void matrix_vectorELL_block(const uint32_t num_rows, const uint32_t testPoint, 
			const uint32_t* num_cols_per_row_vec, 
			const uint32_t* block_data_bias_vec,    
			const uint32_t *J,
 			const double *V, const double *x, double *y,
			const bool CACHE, const uint32_t rodr_blocks, const uint32_t* part_boundary_d,
			const bool tex=false)
{
	uint32_t ELL_blocks = ceil((double) num_rows/ELL_threadSize);
	//printf("ELL_blocks is %d\n", ELL_blocks);
	//bind_x(x);
	
	if(rodr_blocks > 0){
		if(CACHE){	
			ELL_cached_kernel_rodr<<<rodr_blocks, ELL_threadSize>>>(num_cols_per_row_vec, 
					block_data_bias_vec,
					J, V, x, y, part_boundary_d);
			gpuErrchk( hipPeekAtLastError() );
		} else {
			if(testPoint > 0){
				ELL_kernel_rodr_test<<<rodr_blocks, ELL_threadSize>>>(num_cols_per_row_vec, 
						block_data_bias_vec,
						J, V, x, y, part_boundary_d, testPoint);
			} else {
				ELL_kernel_rodr<<<rodr_blocks, ELL_threadSize>>>(num_cols_per_row_vec, 
						block_data_bias_vec,
						J, V, x, y, part_boundary_d);
			}
			
		}

	}else{
		ELL_kernel_block<<<ELL_blocks, ELL_threadSize>>>(num_rows, num_cols_per_row_vec, 
			block_data_bias_vec, J, V, x,y);
	}
	if(tex == true)

		
	//unbind_x(x);
	
}

void matrix_vectorCOO(const uint32_t num_nozeros_compensation, uint32_t *I, uint32_t *J, double *V, double *x_d, double *y_d)
{
	uint32_t interval_size2;
	interval_size2=ceil(((double) num_nozeros_compensation)/(block_size*thread_size/WARP_SIZE));//for data with 2 million elements, we have interval size 200	
	COO_atomic(num_nozeros_compensation, interval_size2, I, J, V, x_d, y_d);
}

void matrix_vectorHYB(matrixHYB_S_d* inputMatrix, double* vector_in_d,
		double* vector_out_d, cb_s cb, const uint32_t testPoint,
		const uint32_t part_size, const uint32_t* part_boundary_d, 
		bool tex=false)
{
	uint32_t dimension = inputMatrix->dimension;
	uint32_t ELL_width = inputMatrix->ELL_width;
	uint32_t totalNumCOO = inputMatrix->totalNumCOO;
	uint32_t* col_d = inputMatrix->col_d;
	uint32_t* I_COO_d = inputMatrix->I_COO_d;
	uint32_t* J_COO_d = inputMatrix->J_COO_d;
	double* V_d = inputMatrix->V_d;
	double* V_COO_d = inputMatrix->V_COO_d;
	uint32_t* ELL_block_bias_vec_d = inputMatrix->ELL_block_bias_vec_d;
	uint32_t* ELL_block_cols_vec_d = inputMatrix->ELL_block_cols_vec_d;
	size_t offset = 0;
	if(tex==true)
		hipBindTexture(&offset, texInput, vector_in_d, sizeof(double)*dimension);	
	if(!cb.BLOCK){
		matrix_vectorELL(dimension, dimension, ELL_width, col_d,V_d,
				vector_in_d, vector_out_d, false, 0, NULL);
	} else {
		if(cb.RODR){
			matrix_vectorELL_block(dimension, testPoint, ELL_block_cols_vec_d, 
					ELL_block_bias_vec_d,
					col_d,V_d, vector_in_d, vector_out_d,
					cb.CACHE, part_size, part_boundary_d);
		}
		else{
			matrix_vectorELL_block(dimension, testPoint, ELL_block_cols_vec_d, 
					ELL_block_bias_vec_d,
					col_d, V_d, 
					vector_in_d, vector_out_d,
					false, 0, NULL);
		}
	}

	if (totalNumCOO > 0) matrix_vectorCOO(totalNumCOO, I_COO_d, J_COO_d, V_COO_d, 
			vector_in_d, vector_out_d);

	if(tex==true)
		hipUnbindTexture(texInput);
}
