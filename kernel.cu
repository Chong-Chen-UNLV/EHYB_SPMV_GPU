#include "hip/hip_runtime.h"
#include "kernel.h"

#define FULL_MASK 0xffffffff
#define BASE 262144 //1024*1024

#define block_size 512	
#define thread_size 512
#define block_size2 16
#define thread_size2 512
#define WARP_SIZE 32

/*kernel function for initialize*/
__global__ void kernelInitialize(const int num, double *x)
{
	int idx=blockDim.x * blockIdx.x+ threadIdx.x;
	
	for (int n=idx;n<num;n+=BASE) x[n]=0;
}

__global__ void kernelInitializeAll(const int num, double *pk, double *bp, double *x, double *zk, const double *vector_in)
{
	int idx=blockDim.x * blockIdx.x+ threadIdx.x;
	double temp;
	for (int n=idx;n<num;n+=BASE) 
	{
		temp=zk[n];
		pk[n]=temp;
		bp[n]=0;
		x[n]=0;
	}
}

__global__ void kernelInitializeR(const int num,double *rk, const double *vector_in)
{
	int idx=blockDim.x * blockIdx.x+ threadIdx.x;
	double temp;
	for (int n=idx;n<num;n+=BASE) 
	{
		temp=vector_in[n];
		rk[n]=temp;
	}
}

//first version implementation,
//concern about performance loss from inbalance between blocks 
__global__ void kernelER(const int numOfRowER,
			const int* widthVecER_d, 
			const int* biasVecER_d,  
			const int* rowVecER_d;
			const int* colER_d, 
			const double *valER_d, const double * x, double * y)
{
	int width;  
	int bias;
	int dataIdx;
	int row, col;	
	double val;
	uint32_t idx = blockDim.x*blockIdx.x+threadIdx.x;
	uint32_t warpIdx = idx>>5;
	row = rowVecER_d[idx];
	if(row < numOfRowER){
		width = widthVecER_d[warpIdx];//cache will work when every threads read same global address
		bias = biasVecER_d[warpIdx];
		double dot = 0;
		for(int n=0; n < width; ++n){
			dataIdx = bias + warpSize*n + threadIdx.x;
			col=colER_d[dataIdx];
			val=valER_d[dataIdx];
			dot += val* x[col];
		}
		y[row]+=dot;
	}
}

__global__ void kernelCachedBlockedELL_test(const int* widthVecBlockELL_d,
		const int* vecBias,  
		const int *indices, const double *data, const double * x,
		double * y,
		const int* partBoundary,
		const int testPoint)
{
	int partIdx = blockIdx.x; 
	int xIdx = threadIdx.x;
	__shared__ volatile double cachedVec[vectorCacheSize];  
	__shared__ volatile int sharedBias[warpPerBlock];  
	__shared__ volatile int sharedWidth[warpPerBlock];  
	int vecStart = partBoundary[blockIdx.x];
	int vecEnd = partBoundary[blockIdx.x + 1];
	int warpLane = xIdx - (xIdx<<5)>>5; //xIdx%32 = xIdx - (xIdx/32)*32)
	int row = 0;

	for (int i = xIdx; i < vectorCacheSize; i += threadELL){
		cachedVec[i] = x[i + vecStart];
	}
	if(xIdx < blockPerPart){
		sharedBias[xIdx] = vecBias[blockPerPart*partIdx + xIdx];	
		sharedWidth[xIdx] = vecWidth[blockPerPart*partIdx + xIdx];	
	}
	double val, dot;
	int blockId, dataIdx; 
	int col;
	int bias, width;

	#pragma unroll
	for(int i = 0; i < memPerThread/elementSize; ++i){//the thread is step with stride ELL_threadSize
		dot = 0;
		//each iteration go through (1024/warpSize)=32 blocks in blockELL format, which is i >> 5
		//the warpIdx is xIdx<<4
		bias = sharedBias[i>>5 + xIdx<<5]; 
		width = sharedWidth[i>>5 + xIdx<<5];
		row = i*threadELL + vecStart + xIdx;
		if(row < vec_end){
			for(int n=0; n< width; ++n){
				dataIdx = bias + warpSize*n + warpLane;//however the data storage is stride with block_rowSize
				val=data[dataIdx];
				col = indices[dataIdx]- vecStart;
				//if(col < 0 || col >= vector_cache_size) 
				//	dot += val*cached_vec[col];
				//else
				if(row == testPoint)
					dot += val*cached_vec[col] - 1 + 0.999;
				else
					dot += val*cachedVec[col];
			}
			//if(row == testPoint)
			//	y[row] = dot+0.01;
			//else 
			y[row] = dot;
		}
	}		
}

__global__ void kernelCachedBlockedELL(const int* widthVecBlockELL_d,
		const int* vecBias,  
		const int *indices, const double *data, const double * x,
		double * y,
		const int* partBoundary)
{
	int partIdx = blockIdx.x; 
	int xIdx = threadIdx.x;
	__shared__ volatile double cachedVec[vectorCacheSize];  
	__shared__ volatile int sharedBias[warpPerBlock];  
	__shared__ volatile int sharedWidth[warpPerBlock];  
	int vecStart = partBoundary[blockIdx.x];
	int vecEnd = partBoundary[blockIdx.x + 1];
	int warpLane = xIdx - (xIdx<<5)>>5; //xIdx%32 = xIdx - (xIdx/32)*32)
	int row = 0;

	for (int i = xIdx; i < vectorCacheSize; i += threadELL){
		cachedVec[i] = x[i + vecStart];
	}
	if(xIdx < blockPerPart){
		sharedBias[xIdx] = vecBias[blockPerPart*partIdx + xIdx];	
		sharedWidth[xIdx] = vecWidth[blockPerPart*partIdx + xIdx];	
	}
	double val, dot;
	int blockId, dataIdx; 
	int col;
	int bias, width;

	#pragma unroll
	for(int i = 0; i < memPerThread/elementSize; ++i){//the thread is step with stride ELL_threadSize
		dot = 0;
		//each iteration go through (1024/warpSize)=32 blocks in blockELL format, which is i >> 5
		//the warpIdx is xIdx<<4
		bias = sharedBias[i>>5 + xIdx<<5]; 
		width = sharedWidth[i>>5 + xIdx<<5];
		row = i*threadELL + vecStart + xIdx;
		if(row < vec_end){
			for(int n=0; n< width; ++n){
				dataIdx = bias + warpSize*n + warpLane;//however the data storage is stride with block_rowSize
				val=data[dataIdx];
				col = indices[dataIdx]- vecStart;
				//if(col < 0 || col >= vector_cache_size) 
				//	dot += val*cached_vec[col];
				//else
				//if(row == testPoint)
				//	dot += val*cached_vec[col] - 1 + 0.999;
				//else
				dot += val*cachedVec[col];
			}
			//if(row == testPoint)
			//	y[row] = dot+0.01;
			//else 
			y[row] = dot;
		}
	}		
}


//y=x+gamak*y
__global__ void kernelMyxpy(const int dimension, double gamak, const double *x, double *y)
{
	int idx=blockDim.x*blockIdx.x+threadIdx.x;
	int n=idx;
	while(n<dimension){
		y[n]=x[n]+gamak*y[n];
		n=n+BASE;
	}
}
	extern "C"
void initialize_all(const int dimension, double *pk_d, double *bp_d, double *x, double *zk, const double *vector_in_d)
{
	kernelInitializeAll<<<block_size,thread_size>>>(dimension, pk_d, bp_d, x, zk, vector_in_d);
}

void initialize_bp(int num, double *x)
{
	kernelInitialize<<<block_size,thread_size>>>(num,x);
}

void initialize_r(int num, double *rk, double *vector_in)
{
	kernelInitializeR<<<block_size,thread_size>>>(num,rk,vector_in);
}
void myxpy(const int dimension, double gamak, const double *x, double *y)
{
	kernelMyxpy<<<block_size,thread_size>>>(dimension,gamak,x,y);
}

void initialDeviceArray(int num, double *x)
{
	kernelInitialize<<<512,512>>>(num,x);
}



void matrixVectorBlockELL(const int nParts, const int testPoint, 
		const int* widthVecBlockELL_d, 
		const int* biasVecBLockELL_d,    
		const int* colBlockELL_d,
		const double* valBlockELL_d, 
		const int* partBoundary_d,
		const double *x_d, double *y_d)
{


	
	//printf("ELL_blocks is %d\n", ELL_blocks);
	if(!CACHE){	
		printf("only examine cached implementation now\n");
		exit(0);	
		//gpuErrchk( hipPeekAtLastError() );
	} else {
		if(testPoint >= 0){
			kernelCachedBlockedELL_test<<<nParts, threadELL>>>(widthVecBlockELL_d,
					biasVecBLockELL_d,  
					colBlockELL_d, valBlockELL_d, 
					x_d,
					y_d,
					partBoundary_d,
					testPoint);
		} else {
			kernelCachedBlockedELL<<<nParts, threadELL>>>(widthVecBlockELL_d,
					biasVecBLockELL_d,  
					colBlockELL_d, valBlockELL_d, 
					x_d,
					y_d,
					partBoundary_d);
		}

	}
}

void matrixVectorER(const int numOfRowER, 
		const int* rowVecER_d, const int* biasVecER_d, 
		const int* widthVecER_d, 
		const int* colER_d, const double* valER_d,
		const double* vectorIn_d, const double* vectorOut_d)
{

	int blockSizeLocal;
	blockSizeLocal=ceil(((double) numOfRowER)/threadSizeELL);//for data with 2 million elements, we have interval size 200
	kernelER<<<blockSizeLocal, threadSizeER>>>(numOfRowER, 
			rowVecER_d, 
			biasVecER_d, 
			widthVecER_d,
			colER_d, 
			valER_d, 
			vectorIn_d, 
			vectorOut_d);

}

void matrixVectorEHYB(matrixEHYB* inputMatrix_d, double* vectorIn_d,
		double* vectorOut_d, const int testPoint)
{

	matrixVectorBlockELL(inputMatrix_d->nParts, 
			testPoint,
			inputMatrix_d->widthVecBlockELL,
			inputMatrix_d->biasVecBLockELL,  
			inputMatrix_d->colBlockELL, 
			inputMatrix_d->valBlockELL, 
			inputMatrix_d->partBoundary,
			vectorIn_d,
			vectorOut_d);
	
	matrixVectorER(numOfRowER, inputMatrix_d->rowVecER, 
			inputMatrix_d->widthVecER,
			inputMatrix_d->biasVecER,
			inputMatrix_d->colER, 
			inputMatrix_d->valER,
			vectorIn_d, vectorOut_d);

}
