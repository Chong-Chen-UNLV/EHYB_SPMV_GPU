#include "hip/hip_runtime.h"
#include "kernel.h"

#define FULL_MASK 0xffffffff
#define BASE 262144 //1024*1024

#define block_size 512	
#define thread_size 512
#define block_size2 16
#define thread_size2 512
#define warpSize 32

/*kernel function for initialize*/
__global__ void kernelInitialize(const int num, double *x)
{
	int idx=blockDim.x * blockIdx.x+ threadIdx.x;
	
	for (int n=idx;n<num;n+=BASE) x[n]=0;
}

__global__ void kernelInitializeAll(const int num, double *pk, double *bp, double *x, double *zk, const double *vector_in)
{
	int idx=blockDim.x * blockIdx.x+ threadIdx.x;
	double temp;
	for (int n=idx;n<num;n+=BASE) 
	{
		temp=zk[n];
		pk[n]=temp;
		bp[n]=0;
		x[n]=0;
	}
}

__global__ void kernelInitializeR(const int num,double *rk, const double *vector_in)
{
	int idx=blockDim.x * blockIdx.x+ threadIdx.x;
	double temp;
	for (int n=idx;n<num;n+=BASE) 
	{
		temp=vector_in[n];
		rk[n]=temp;
	}
}

//first version implementation,
//concern about performance loss from inbalance between blocks 
__global__ void kernelER(const int numOfRowER,
			const int* rowVecER,
			const int* biasVecER,  
			const int* widthVecER, 
			const int* colER, 
			const double *valER, const double * x, double * y)
{
	int width;  
	int bias;
	int dataIdx;
	int row, col;	
	double val;
	uint32_t idx = blockDim.x*blockIdx.x+threadIdx.x;
	uint32_t warpIdx = idx>>5;
	uint32_t warpLane = threadIdx.x- ((threadIdx.x>>5)<<5);
	if(idx < numOfRowER){
		row = rowVecER[idx];
		width = widthVecER[warpIdx];//cache will work when every threads read same global address
		bias = biasVecER[warpIdx];
		double dot = 0;
		for(int n=0; n < width; ++n){
			dataIdx = bias + warpLane + warpSize*n ;
			col=colER[dataIdx];
			val=valER[dataIdx];
			dot += val* x[col];
		}
		y[row]+=dot;
	}
}

__global__ void kernelCachedBlockedELL_test(const int* widthVecBlockELL,
		const int* biasVecBlockELL,  
		const int16_t *colBlockELL, 
		const double *valBlockELL, 
		const double * x,
		double * y,
		const int* partBoundary,
		const int testPoint)
{
	int partIdx = blockIdx.x; 
	int xIdx = threadIdx.x;
	__shared__ volatile double cachedVec[vectorCacheSize];  
	int vecStart = partBoundary[blockIdx.x];
	int vecEnd = partBoundary[blockIdx.x + 1];
	int warpLane = xIdx - ((xIdx>>5)<<5); //xIdx%32 = xIdx - (xIdx/32)*32)
	int row = 0;
	int blockStartIdx = blockPerPart*partIdx;	
	for (int i = xIdx; i < vectorCacheSize; i += threadELL){
		cachedVec[i] = x[i + vecStart];
	}
	
	__syncthreads();
	double val, dot;
	int dataIdx; 
	int col;
	int biasIdx, bias, width;

	#pragma unroll
	for(int i = 0; i < loopInKernel; ++i){//the thread is step with stride threadELL
		dot = 0;
		//each iteration go through (1024/warpSize)=32 blocks in blockELL format, which is i >> 5
		//the warpIdx is xIdx>>5
		row = i*threadELL + vecStart + xIdx;
		if(row < vecEnd){
			biasIdx = i*warpPerBlock + (xIdx>>5) + blockStartIdx;
			bias = biasVecBlockELL[biasIdx]; 
			width = widthVecBlockELL[biasIdx];
			for(int n=0; n< width; ++n){
				dataIdx = bias + warpSize*n + warpLane;//however the data storage is stride with block_rowSize
				val= valBlockELL[dataIdx];
				col = colBlockELL[dataIdx];
				if(row == testPoint)
					dot += val*cachedVec[col] - 1 + 0.999;
				else
					dot += val*cachedVec[col];
			}
			//if(row == testPoint)
			//	y[row] = dot+0.01;
			//else 
			y[row] = dot;
		}
	}		
}

__global__ void kernelCachedBlockedELL(
		//int16_t* biasIdxBlock,
		const int* widthVecBlockELL,
		const int* biasVecBlockELL,  
		const int16_t *colBlockELL, 
		const double *valBlockELL, 
		const double * x,
		double * y,
		const int* partBoundary)
{
	int partIdx = blockIdx.x; 
	int xIdx = threadIdx.x;
	__shared__ volatile double cachedVec[vectorCacheSize];  
	__shared__ int biasIdxBlock; 
	//__shared__ volatile int sharedBias[blockPerPart];  
	//__shared__ volatile int sharedWidth[blockPerPart];  
	int vecStart = partBoundary[blockIdx.x];
	int vecEnd = partBoundary[blockIdx.x + 1];
	int warpLane = xIdx - ((xIdx>>5)<<5); //xIdx%32 = xIdx - (xIdx/32)*32)
	int warpIdx = (xIdx/32);
	int row = 0;
	int biasIdxWarp;
	int blockStartIdx = blockPerPart*partIdx;	
	for (int i = xIdx; i < vectorCacheSize; i += threadELL){
		cachedVec[i] = x[i + vecStart];
	}
	//if(xIdx < blockPerPart){
	//	sharedBias[xIdx] = biasVecBlockELL[blockStartIdx + xIdx];	
	//	sharedWidth[xIdx] = widthVecBlockELL[blockStartIdx+ xIdx];	
	//}
	if(xIdx == 0) biasIdxBlock = warpPerBlock; 
	biasIdxWarp = warpIdx;
	__syncthreads();
	double val, dot;
	int dataIdx; 
	int col;
	int bias, width;
	#pragma unroll
	for(int i = 0; i < loopInKernel; ++i){//the thread is step with stride threadELL
		dot = 0;
		row = warpLane + biasIdxWarp*warpSize + vecStart;
		if(row < vecEnd){
			bias = biasVecBlockELL[biasIdxWarp + blockStartIdx]; 
			width = widthVecBlockELL[biasIdxWarp + blockStartIdx];
			for(int n=0; n< width; ++n){
				dataIdx = bias + warpSize*n + warpLane;//however the data storage is stride with block_rowSize
				val= valBlockELL[dataIdx];
				col = colBlockELL[dataIdx];
				dot += val*cachedVec[col];
			}
			//if(row == testPoint)
			//	y[row] = dot+0.01;
			//else 
			y[row] = dot;
		}
		if(warpLane == 0)
			biasIdxWarp = atomicAdd(&biasIdxBlock, 1); 
		biasIdxWarp = __shfl_sync(FULL_MASK, biasIdxWarp, 0);
	 	__syncwarp();	
	}
}

__global__ void kernelCachedBlockedELL_NC(
		//int16_t* biasIdxBlock,
		const int* widthVecBlockELL,
		const int* biasVecBlockELL,  
		const int16_t *colBlockELL, 
		const double *valBlockELL, 
		const double * x,
		double * y,
		const int* partBoundary)
{
	int partIdx = blockIdx.x; 
	int xIdx = threadIdx.x;
	//__shared__ volatile double cachedVec[vectorCacheSize];  
	__shared__ int biasIdxBlock; 
	//__shared__ volatile int sharedBias[blockPerPart];  
	//__shared__ volatile int sharedWidth[blockPerPart];  
	int vecStart = partBoundary[blockIdx.x];
	int vecEnd = partBoundary[blockIdx.x + 1];
	int warpLane = xIdx - ((xIdx>>5)<<5); //xIdx%32 = xIdx - (xIdx/32)*32)
	int warpIdx = (xIdx/32);
	int row = 0;
	int biasIdxWarp;
	int blockStartIdx = blockPerPart*partIdx;	
	//for (int i = xIdx; i < vectorCacheSize; i += threadELL){
	//	cachedVec[i] = x[i + vecStart];
	//}
	//if(xIdx < blockPerPart){
	//	sharedBias[xIdx] = biasVecBlockELL[blockStartIdx + xIdx];	
	//	sharedWidth[xIdx] = widthVecBlockELL[blockStartIdx+ xIdx];	
	//}
	if(xIdx == 0) biasIdxBlock = warpPerBlock; 
	biasIdxWarp = warpIdx;
	__syncthreads();
	double val, dot;
	int dataIdx; 
	int col;
	int bias, width;
	#pragma unroll
	for(int i = 0; i < loopInKernel; ++i){//the thread is step with stride threadELL
		dot = 0;
		row = warpLane + biasIdxWarp*warpSize + vecStart;
		if(row < vecEnd){
			bias = biasVecBlockELL[biasIdxWarp + blockStartIdx]; 
			width = widthVecBlockELL[biasIdxWarp + blockStartIdx];
			for(int n=0; n< width; ++n){
				dataIdx = bias + warpSize*n + warpLane;//however the data storage is stride with block_rowSize
				val= valBlockELL[dataIdx];
				col = colBlockELL[dataIdx];
				dot += val*x[vecStart + col];
			}
			//if(row == testPoint)
			//	y[row] = dot+0.01;
			//else 
			y[row] = dot;
		}
		if(warpLane == 0)
			biasIdxWarp = atomicAdd(&biasIdxBlock, 1); 
		biasIdxWarp = __shfl_sync(FULL_MASK, biasIdxWarp, 0);
	 	__syncwarp();	
	}
}


//y=x+gamak*y
__global__ void kernelMyxpy(const int dimension, double gamak, const double *x, double *y)
{
	int idx=blockDim.x*blockIdx.x+threadIdx.x;
	int n=idx;
	while(n<dimension){
		y[n]=x[n]+gamak*y[n];
		n=n+BASE;
	}
}

extern "C"
void initialize_all(const int dimension, double *pk_d, double *bp_d, double *x, double *zk, const double *vector_in_d)
{
	kernelInitializeAll<<<block_size,thread_size>>>(dimension, pk_d, bp_d, x, zk, vector_in_d);
}

void initialize_bp(int num, double *x)
{
	kernelInitialize<<<block_size,thread_size>>>(num,x);
}

void initialize_r(int num, double *rk, double *vector_in)
{
	kernelInitializeR<<<block_size,thread_size>>>(num,rk,vector_in);
}
void myxpy(const int dimension, double gamak, const double *x, double *y)
{
	kernelMyxpy<<<block_size,thread_size>>>(dimension,gamak,x,y);
}

void initialDeviceArray(int num, double *x)
{
	kernelInitialize<<<512,512>>>(num,x);
}



void matrixVectorBlockELL(const int nParts, const int testPoint, 
		//int16_t* biasIdxBlock_d,
		const int* widthVecBlockELL_d, 
		const int* biasVecBlockELL_d,    
		const int16_t* colBlockELL_d,
		const double* valBlockELL_d, 
		const int* partBoundary_d,
		const double *x_d, double *y_d)
{

		if(testPoint >= 0){
			kernelCachedBlockedELL_test<<<nParts, threadELL>>>(widthVecBlockELL_d,
					biasVecBlockELL_d,  
					colBlockELL_d, valBlockELL_d, 
					x_d,
					y_d,
					partBoundary_d,
					testPoint);
		} else {
			kernelCachedBlockedELL<<<nParts, threadELL>>>(
					//biasIdxBlock_d,
					widthVecBlockELL_d,
					biasVecBlockELL_d,  
					colBlockELL_d, valBlockELL_d, 
					x_d,
					y_d,
					partBoundary_d);
		}

}




void matrixVectorER(const int numOfRowER, 
		const int* rowVecER_d, const int* biasVecER_d, 
		const int* widthVecER_d, 
		const int* colER_d, const double* valER_d,
		const double* vectorIn_d, double* vectorOut_d)
{

	int blockSizeLocal;
	blockSizeLocal=ceil(((double) numOfRowER)/threadELL);//for data with 2 million elements, we have interval size 200
	kernelER<<<blockSizeLocal, threadELL>>>(numOfRowER, 
			rowVecER_d, 
			biasVecER_d, 
			widthVecER_d,
			colER_d, 
			valER_d, 
			vectorIn_d, 
			vectorOut_d);

}

void matrixVectorEHYB_NC(matrixEHYB* inputMatrix_d, 
		//int16_t* biasIdxBlock_d, 
		double* vectorIn_d,
		double* vectorOut_d, const int testPoint)
{

	kernelCachedBlockedELL_NC<<<inputMatrix_d->nParts, threadELL>>>(
			//biasIdxBlock_d,
			inputMatrix_d->widthVecBlockELL,
			inputMatrix_d->biasVecBlockELL,  
			inputMatrix_d->colBlockELL, 
			inputMatrix_d->valBlockELL, 
			vectorIn_d,
			vectorOut_d,
			inputMatrix_d->partBoundary);

	
	matrixVectorER(inputMatrix_d->numOfRowER, inputMatrix_d->rowVecER, 
			inputMatrix_d->biasVecER,
			inputMatrix_d->widthVecER,
			inputMatrix_d->colER, 
			inputMatrix_d->valER,
			vectorIn_d, vectorOut_d);

}

void matrixVectorEHYB(matrixEHYB* inputMatrix_d, 
		//int16_t* biasIdxBlock_d, 
		double* vectorIn_d,
		double* vectorOut_d, const int testPoint)
{

	matrixVectorBlockELL(inputMatrix_d->nParts, 
			testPoint,
			//biasIdxBlock_d,
			inputMatrix_d->widthVecBlockELL,
			inputMatrix_d->biasVecBlockELL,  
			inputMatrix_d->colBlockELL, 
			inputMatrix_d->valBlockELL, 
			inputMatrix_d->partBoundary,
			vectorIn_d,
			vectorOut_d);
	
	matrixVectorER(inputMatrix_d->numOfRowER, inputMatrix_d->rowVecER, 
			inputMatrix_d->biasVecER,
			inputMatrix_d->widthVecER,
			inputMatrix_d->colER, 
			inputMatrix_d->valER,
			vectorIn_d, vectorOut_d);

}
