#include "hip/hip_runtime.h"
#include "kernel.h"
#include "solver.h"
#include "convert.h"
#include "reordering.h"
#include "test.h"

static void cudaMallocTransDataEHYB(matrixEHYB* localMatrix, matrixEHYB* localMatrix_d, 
		const int sizeBlockELL, const int sizeER){


	localMatrix_d->dimension = localMatrix->dimension;
	localMatrix_d->numOfRowER = localMatrix->numOfRowER;
	localMatrix_d->nParts = localMatrix->nParts;
	int blockNumER = ceil(((float) localMatrix->numOfRowER)/warpSize);

    hipMalloc((void **) &(localMatrix_d->biasVecBlockELL), localMatrix->nParts*blockPerPart*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->widthVecBlockELL), localMatrix->nParts*blockPerPart*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->partBoundary), (localMatrix->nParts+1)*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->valBlockELL), sizeBlockELL*sizeof(double));
    hipMalloc((void **) &(localMatrix_d->colBlockELL), sizeBlockELL*sizeof(int));

    hipMalloc((void **) &(localMatrix_d->rowVecER), localMatrix_d->numOfRowER*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->biasVecER), blockNumER*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->widthVecER), blockNumER*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->colER), sizeER*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->valER), sizeER*sizeof(double));

    hipMemcpy(localMatrix_d->biasVecBlockELL, localMatrix->biasVecBlockELL, localMatrix->nParts*blockPerPart*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->widthVecBlockELL, localMatrix->widthVecBlockELL, localMatrix->nParts*blockPerPart*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->partBoundary, localMatrix->partBoundary, (localMatrix->nParts+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->valBlockELL, localMatrix->valBlockELL, sizeBlockELL*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->colBlockELL, localMatrix->colBlockELL, sizeBlockELL*sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(localMatrix_d->rowVecER, localMatrix->rowVecER, localMatrix_d->numOfRowER*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->biasVecER, localMatrix->biasVecER, blockNumER*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->widthVecER, localMatrix->widthVecER, blockNumER*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->colER, localMatrix->colER, sizeER*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->valER, localMatrix->valER, sizeER*sizeof(double), hipMemcpyHostToDevice);
}
extern "C"
void solverGPuUnprecondEHYB(matrixCOO* localMatrix, 
		const double *vectorIn, double *vectorOut,  
		const int MAXIter, int *realIter)
{
	//This function treat y as input and x as output, (solve the equation Ax=y) y is the vector we already known, x is the vector we are looking for
	double dotp0,dotr0,dotr1,doth;

	int sizeBlockELL, sizeER;
	int dimension = localMatrix->dimension;
	int totalNum = localMatrix->totalNum;
	matrixEHYB localMatrixEHYB, localMatrixEHYB_d;

	COO2EHYB(localMatrix, 
			&localMatrixEHYB,
			&sizeBlockELL,
			&sizeER);

	cudaMallocTransDataEHYB(&localMatrixEHYB,
			&localMatrixEHYB_d, 
			sizeBlockELL,
			sizeER);
	printf("sizeER is %d\n", sizeER);
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	double *bp_d, *pk_d, *rk_d, *vectorOut_d;
	size_t size1 = dimension*sizeof(double);
	hipMalloc((void **) &bp_d,size1);
	hipMalloc((void **) &pk_d,size1);
	hipMalloc((void **) &rk_d,size1);
	hipMalloc((void **) &vectorOut_d,size1);
	//double *x=(double *) malloc(size1);
	double threshold=0.0000001;
	int iter=0;
	double const1 = 1.0;
	double error, alphak, _alphak, gamak;
	error=1000;
	//initialize
	doth=0;
    hipMemcpy(pk_d, vectorIn, dimension*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(rk_d, vectorIn, dimension*sizeof(double), hipMemcpyHostToDevice);
	for (int i=0;i<dimension;i++) {
		doth=doth+vectorIn[i]*vectorIn[i];
	}
	struct timeval start1, end1;
	double *bp=(double *) malloc(size1);
	double *bp_g =(double *) malloc(size1);
	double *pk=(double *) malloc(size1);
	double *rk=(double *) malloc(size1);

	//double *x=(double *) malloc(size1);
	error=1000;
	//initialize
	for (int i=0;i<dimension;i++)
	{
		pk[i]=vectorIn[i];
		rk[i]=vectorIn[i];
		vectorOut[i]=0;
		bp[i]=0;
		bp_g[i]=0;
	}
	gettimeofday(&start1, NULL);
	while (error>threshold&&iter<MAXIter){
		dotp0=0;
		dotr0=0;
		dotr1=0;
		int errorIdx = 0;
		double compareError;
		hipMemset(bp_d, 0, size1);
		matrixVectorEHYB(&localMatrixEHYB_d, pk_d, bp_d, -1);
		hipblasDdot(handle,dimension,bp_d,1,pk_d,1,&dotp0);
		hipblasDdot(handle,dimension,rk_d,1,rk_d,1,&dotr0);
			
		alphak=dotr0/dotp0;
		_alphak = -alphak;
		
		hipblasDaxpy(handle,dimension,&alphak,pk_d,1,vectorOut_d,1);
		hipblasDaxpy(handle,dimension,&_alphak,bp_d,1,rk_d,1);
		hipblasDdot(handle,dimension,rk_d,1,rk_d,1,&dotr1);
		
		gamak=dotr1/dotr0;

		hipblasDscal(handle,dimension,&gamak, pk_d,1);
		hipblasDaxpy(handle,dimension,&const1, rk_d, 1, pk_d, 1);
		
		//printf("at iter %d, alphak is %f, gamak is %f\n",iter, alphak,gamak);
		error=sqrt(dotr1)/sqrt(doth);
		//error_track[iter]=error;
		//printf("error at %d is %f\n",iter, error);
		iter++;
	}
	hipMemcpy(vectorOut, vectorOut_d, dimension*sizeof(double), hipMemcpyDeviceToHost);
	gettimeofday(&end1, NULL);	
	double timeByMs=((end1.tv_sec * 1000000 + end1.tv_usec)-(start1.tv_sec * 1000000 + start1.tv_usec))/1000;
	printf("iter is %d, time is %f ms, GPU Gflops is %f, under estimate flops is %f\n ",iter, timeByMs, 
			(1e-9*(totalNum*2+13*dimension)*1000*iter)/timeByMs, (1e-9*(totalNum*2)*1000*iter)/timeByMs);
	hipFree(localMatrixEHYB_d.valER);
	hipFree(localMatrixEHYB_d.colER);
	hipFree(localMatrixEHYB_d.biasVecER);
	hipFree(localMatrixEHYB_d.widthVecER);
	hipFree(localMatrixEHYB_d.rowVecER);
	hipFree(localMatrixEHYB_d.biasVecBlockELL);
	hipFree(localMatrixEHYB_d.widthVecBlockELL);
	hipFree(localMatrixEHYB_d.colBlockELL);
	hipFree(localMatrixEHYB_d.valBlockELL);
	hipFree(localMatrixEHYB_d.partBoundary);
}

void solverGPuUnprecondCUSPARSE(matrixCOO* localMatrix, 
		const double *vector_in, double *vector_out,  
		const int MAXIter, int *realIter,  const cb_s cb,
		const int partSize, const int* partBoundary)
{
	//exampine the performance using cusparse library functions with
	//CSR format
	//double dotp0,dotr0,dotr1,doth;
	double dotp0,dotr0,dotr1,doth;
	int dimension, totalNum; 
    int *rowIdx, *J; 
    double* V;
    dimension = localMatrix->dimension; 
    totalNum = localMatrix->totalNum; 

	rowIdx = localMatrix->rowIdx; 
    J = localMatrix->J;
    V = localMatrix->V;
	
	int* col_d;
	int* rowIdx_d;
	double *V_d;
	hipblasHandle_t handleBlas;
	hipblasCreate(&handleBlas);
	hipsparseHandle_t handleSparse;
	hipsparseCreate(&handleSparse);

	double *bp_d, *pk_d, *rk_d, *vector_out_d;
	size_t size1=dimension*sizeof(double);
	hipMalloc((void **) &bp_d,size1);
	hipMalloc((void **) &pk_d,size1);
	hipMalloc((void **) &rk_d,size1);
	hipMalloc((void **) &rowIdx_d,size1);
	hipMalloc((void **) &vector_out_d,size1);
	hipMalloc((void **) &col_d,totalNum*sizeof(int));
	hipMalloc((void **) &V_d,totalNum*sizeof(double));
	//double *x=(double *) malloc(size1);
	double threshold=0.0000001;
	int iter=0;
	double const1 = 1.0;
	double error, alphak, _alphak, gamak;
	error=1000;
	//initialize
	doth=0;
    hipMemcpy(pk_d, vector_in, dimension*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(rk_d, vector_in, dimension*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(rowIdx_d, rowIdx, dimension*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(col_d, J, totalNum*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(V_d, V, totalNum*sizeof(double), hipMemcpyHostToDevice);
	for (int i=0;i<dimension;i++) {
		doth=doth+vector_in[i]*vector_in[i];
	}
	struct timeval start1, end1;
	double *bp=(double *) malloc(size1);
	double *bp_g =(double *) malloc(size1);
	double *pk=(double *) malloc(size1);
	double *rk=(double *) malloc(size1);

	//double *bp_dt = (double *) malloc(size1);
	//double *pk_dt = (double *) malloc(size1);
	//double *rk_dt = (double *) malloc(size1);
	//double *x=(double *) malloc(size1);
	error=1000;
	//initialize
	for (int i=0;i<dimension;i++)
	{
		pk[i]=vector_in[i];
		rk[i]=vector_in[i];
		vector_out[i]=0;
		bp[i]=0;
	}
	//if BSR doing the format change 
	//cusparseStatus_tcusparseDcsr2gebsr_bufferSize(handle, dir, m, n, descrA, csrValA, csrRowPtrA, 
	//		csrColIndA, rowBlockDim, colBlockDim, pBufferSize);
	hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
	hipsparseMatDescr_t descr = 0;
	int status = hipsparseCreateMatDescr(&descr);
	if (status != HIPSPARSE_STATUS_SUCCESS ) {
		exit(0);	
	}
	hipsparseSetMatType (descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase (descr, HIPSPARSE_INDEX_BASE_ZERO);
	gettimeofday(&start1, NULL);
	double one = 1.0;
	double zero = 0.0;
	while (error>threshold&&iter<MAXIter){
		dotp0=0;
		dotr0=0;
		dotr1=0;
		//int errorIdx = 0;
		//double compareError;
		
		hipMemset(bp_d, 0, size1);
		hipsparseStatus_t smpvStatus = 
		hipsparseDcsrmv(handleSparse,
				transA,
				dimension,
				dimension,
				totalNum,
				&one,
				descr,
				V_d,
				rowIdx_d,
				col_d,
				pk_d,
				&zero,
				bp_d);

		hipblasDdot(handleBlas,dimension,bp_d,1,pk_d,1,&dotp0);
		hipblasDdot(handleBlas,dimension,rk_d,1,rk_d,1,&dotr0);
			
		alphak=dotr0/dotp0;
		_alphak = -alphak;
		
		hipblasDaxpy(handleBlas,dimension,&alphak,pk_d,1,vector_out_d,1);
		hipblasDaxpy(handleBlas,dimension,&_alphak,bp_d,1,rk_d,1);
		hipblasDdot(handleBlas,dimension,rk_d,1,rk_d,1,&dotr1);
		
		gamak=dotr1/dotr0;

		hipblasDscal(handleBlas,dimension,&gamak,pk_d,1);
		hipblasDaxpy(handleBlas,dimension,&const1, rk_d, 1, pk_d, 1);
		
		//printf("at iter %d, alphak is %f, gamak is %f\n",iter, alphak,gamak);
		error=sqrt(dotr1)/sqrt(doth);
		//error_track[iter]=error;
		//printf("error at %d is %f\n",iter, error);
		iter++;
	}
	hipMemcpy(vector_out, vector_out_d, dimension*sizeof(double), hipMemcpyDeviceToHost);
	gettimeofday(&end1, NULL);	
	double timeByMs=((end1.tv_sec * 1000000 + end1.tv_usec)-(start1.tv_sec * 1000000 + start1.tv_usec))/1000;
	printf("iter is %d, time is %f ms, GPU Gflops is %f, under estimate flops is %f\n ",iter, timeByMs, 
			(1e-9*(totalNum*2+13*dimension)*1000*iter)/timeByMs, (1e-9*(totalNum*2)*1000*iter)/timeByMs);

}

