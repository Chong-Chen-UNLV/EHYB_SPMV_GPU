#include "hip/hip_runtime.h"
#include "kernel.h"
#include "spmv.h"
#include "convert.h"
#include "reordering.h"

static void cudaMallocTransDataEHYB(matrixEHYB* localMatrix, matrixEHYB* localMatrix_d, 
		const int sizeBlockELL, const int sizeER){


	localMatrix_d->dimension = localMatrix->dimension;
	localMatrix_d->kernelPerPart = localMatrix->kernelPerPart;
	localMatrix_d->numOfRowER = localMatrix->numOfRowER;
	localMatrix_d->nParts = localMatrix->nParts;
	localMatrix_d->vectorCacheSize = localMatrix->vectorCacheSize;
	int blockNumER = ceil(((double) localMatrix->numOfRowER)/warpSize);
	int warpIdxER = 0;	
	int blockPerPart = (localMatrix->vectorCacheSize/warpSize);
	int vecEleSize = localMatrix->longVecBoundary[localMatrix->nLongVec];

    hipMalloc((void **) &(localMatrix_d->warpIdxER_d), sizeof(int));
    hipMalloc((void **) &(localMatrix_d->outER), localMatrix_d->numOfRowER*sizeof(double));
    hipMalloc((void **) &(localMatrix_d->biasVecBlockELL), localMatrix->nParts*blockPerPart*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->widthVecBlockELL), localMatrix->nParts*blockPerPart*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->partBoundary), (localMatrix->nParts+1)*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->valBlockELL), sizeBlockELL*sizeof(double));
    hipMalloc((void **) &(localMatrix_d->colBlockELL), sizeBlockELL*sizeof(int16_t));

    hipMalloc((void **) &(localMatrix_d->rowVecER), localMatrix_d->numOfRowER*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->biasVecER), blockNumER*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->widthVecER), blockNumER*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->colER), sizeER*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->valER), sizeER*sizeof(double));

	if(localMatrix->nLongVec > 0){
    	hipMalloc((void **) &(localMatrix_d->longVecBoundary), sizeof(int)*localMatrix->nLongVec + 1);
    	hipMalloc((void **) &(localMatrix_d->longVecRow), sizeof(int)*localMatrix->nLongVec);
    	hipMalloc((void **) &(localMatrix_d->longVecCol), sizeof(int)*vecEleSize);
    	hipMalloc((void **) &(localMatrix_d->longVecVal), sizeof(double)*vecEleSize);
	}

    hipMemcpy(localMatrix_d->warpIdxER_d, &warpIdxER, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->biasVecBlockELL, localMatrix->biasVecBlockELL, localMatrix->nParts*blockPerPart*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->widthVecBlockELL, localMatrix->widthVecBlockELL, localMatrix->nParts*blockPerPart*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->partBoundary, localMatrix->partBoundary, (localMatrix->nParts+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->valBlockELL, localMatrix->valBlockELL, sizeBlockELL*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->colBlockELL, localMatrix->colBlockELL, sizeBlockELL*sizeof(int16_t), hipMemcpyHostToDevice);

    hipMemcpy(localMatrix_d->rowVecER, localMatrix->rowVecER, localMatrix_d->numOfRowER*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->biasVecER, localMatrix->biasVecER, blockNumER*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->widthVecER, localMatrix->widthVecER, blockNumER*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->colER, localMatrix->colER, sizeER*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->valER, localMatrix->valER, sizeER*sizeof(double), hipMemcpyHostToDevice);

	if(localMatrix->nLongVec > 0){
    	hipMemcpy(localMatrix_d->longVecBoundary, localMatrix->longVecBoundary, (localMatrix_d->nLongVec+1)*sizeof(int), hipMemcpyHostToDevice);
    	hipMemcpy(localMatrix_d->longVecRow, localMatrix->longVecRow, localMatrix_d->nLongVec*sizeof(int), hipMemcpyHostToDevice);
    	hipMemcpy(localMatrix_d->longVecCol, localMatrix->longVecCol, vecEleSize*sizeof(int), hipMemcpyHostToDevice);
    	hipMemcpy(localMatrix_d->longVecVal, localMatrix->longVecCol, vecEleSize*sizeof(double), hipMemcpyHostToDevice);
	}
}
extern "C"
void spmvGPuEHYB(matrixCOO* localMatrix, 
		const double *vectorIn, double *vectorOut,  
		const int MAXIter, int *realIter)
{
	//This function treat y as input and x as output, (solve the equation Ax=y) y is the vector we already known, x is the vector we are looking for

	int sizeBlockELL, sizeER;
	int dimension = localMatrix->dimension;
	int totalNum = localMatrix->totalNum;
	matrixEHYB localMatrixEHYB, localMatrixEHYB_d;

	COO2EHYB(localMatrix, 
			&localMatrixEHYB,
			&sizeBlockELL,
			&sizeER);

	cudaMallocTransDataEHYB(&localMatrixEHYB,
			&localMatrixEHYB_d, 
			sizeBlockELL,
			sizeER);
	printf("sizeER is %d\n", sizeER);

	double *vectorIn_d, *vectorOut_d;
	int *biasIdxBlock_d;
	if(localMatrix->nParts <= smSize/2){
		hipMalloc((void**) &biasIdxBlock_d, localMatrix->nParts*sizeof(int));
	}
	size_t size1 = dimension*sizeof(double);
	hipMalloc((void **) &vectorOut_d,size1);
	hipMalloc((void **) &vectorIn_d,size1);
	//double *x=(double *) malloc(size1);
	int iter=0;
	struct timeval start1, end1;

	//double *x=(double *) malloc(size1);
	//initialize
	//warm Up
	hipMemcpy(vectorIn_d, vectorIn, dimension*sizeof(double), hipMemcpyHostToDevice);
	for(int i = 0; i < 10; ++i){
		if(localMatrix->nParts <= smSize/2)
		    matrixVectorEHYB_small(&localMatrixEHYB_d, biasIdxBlock_d, vectorIn_d, vectorOut_d);
		else
		    matrixVectorEHYB(&localMatrixEHYB_d, vectorIn_d, vectorOut_d);

	}
	hipMemcpy(vectorOut, vectorOut_d, dimension*sizeof(double), hipMemcpyDeviceToHost);
	gettimeofday(&start1, NULL);
	hipMemcpy(vectorIn_d, vectorIn, dimension*sizeof(double), hipMemcpyHostToDevice);
	while (iter<MAXIter){
		if(localMatrix->nParts <= smSize/2)
		    matrixVectorEHYB_small(&localMatrixEHYB_d, biasIdxBlock_d, vectorIn_d, vectorOut_d);
		else
		    matrixVectorEHYB(&localMatrixEHYB_d, vectorIn_d, vectorOut_d);
		iter++;
	}
	hipMemcpy(vectorOut, vectorOut_d, dimension*sizeof(double), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	gettimeofday(&end1, NULL);	
	double timeByMs=(double (end1.tv_sec * 1000000 + end1.tv_usec)-(start1.tv_sec * 1000000 + start1.tv_usec))/1000;
	printf("iter is %d, time is %f ms, GPU Gflops is %f\n ",iter, timeByMs, 
			(1e-9*(totalNum*2)*1000*iter)/timeByMs );
	hipFree(localMatrixEHYB_d.valER);
	hipFree(localMatrixEHYB_d.colER);
	hipFree(localMatrixEHYB_d.biasVecER);
	hipFree(localMatrixEHYB_d.widthVecER);
	hipFree(localMatrixEHYB_d.rowVecER);
	hipFree(localMatrixEHYB_d.biasVecBlockELL);
	hipFree(localMatrixEHYB_d.widthVecBlockELL);
	hipFree(localMatrixEHYB_d.colBlockELL);
	hipFree(localMatrixEHYB_d.valBlockELL);
	hipFree(localMatrixEHYB_d.partBoundary);
}

void spmvGeneric(matrixCOO* localMatrix, 
		const double *vector_in, double *vector_out,  
		const int MAXIter)
{
	//exampine the performance using cusparse library functions with
	//CSR format
	//double dotp0,dotr0,dotr1,doth;
	int dimension, totalNum; 
    int *rowIdx, *J; 
    double* V;
    dimension = localMatrix->dimension; 
    totalNum = localMatrix->totalNum; 

	rowIdx = localMatrix->rowIdx; 
    J = localMatrix->J;
    V = localMatrix->V;
	
	int* col_d;
	int* rowIdx_d;
	double *V_d;

	double *vector_in_d, *vector_out_d;
	size_t size1=dimension*sizeof(double);
	
	hipMalloc((void **) &rowIdx_d, (dimension+1)*sizeof(double));
	hipMalloc((void **) &vector_out_d,size1);
	hipMalloc((void **) &vector_in_d,size1);
	hipMalloc((void **) &col_d,totalNum*sizeof(int));
	hipMalloc((void **) &V_d,totalNum*sizeof(double));
	//double *x=(double *) malloc(size1);
	int iter=0;
	//double const1 = 1.0;
	//initialize
   	if(hipSuccess != hipMemcpy(rowIdx_d, rowIdx, (dimension+1)*sizeof(int), hipMemcpyHostToDevice)) printf("error1\n");
    if(hipSuccess !=hipMemcpy(col_d, J, totalNum*sizeof(int), hipMemcpyHostToDevice)) printf("error2\n");
    if(hipSuccess !=hipMemcpy(V_d, V, totalNum*sizeof(double), hipMemcpyHostToDevice)) printf("error3\n");
	
	struct timeval start1, end1;
	
	//if BSR doing the format change 
	//cusparseStatus_tcusparseDcsr2gebsr_bufferSize(handle, dir, m, n, descrA, csrValA, csrRowPtrA, 
	//		csrColIndA, rowBlockDim, colBlockDim, pBufferSize);
	hipsparseHandle_t handleSparse;
	hipsparseCreate(&handleSparse);
	hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
	hipsparseSpMatDescr_t descrA = 0;
	hipsparseDnVecDescr_t descrVecIn = 0;
	hipsparseDnVecDescr_t descrVecOut = 0;
	int status = hipsparseCreateDnVec(&descrVecIn,
			dimension,
			vector_in_d,
			HIP_R_32F);
	if (status != HIPSPARSE_STATUS_SUCCESS ) {
		exit(0);	
	}
	status = hipsparseCreateDnVec(&descrVecOut,
			dimension,
			vector_out_d,
			HIP_R_32F);
	if (status != HIPSPARSE_STATUS_SUCCESS ) {
		exit(0);	
	}
	status = hipsparseCreateCsr(&descrA,
			dimension,
			dimension,
			totalNum,
			rowIdx_d,
			col_d,
			V_d,
			HIPSPARSE_INDEX_32I,
			HIPSPARSE_INDEX_32I,
			HIPSPARSE_INDEX_BASE_ZERO,
			HIP_R_32F);
	if (status != HIPSPARSE_STATUS_SUCCESS ) {
		exit(0);	
	}
	double one = 1.0;
	double zero = 0.0;
	size_t buffSize;
	hipsparseStatus_t smpvStatus = 
	hipsparseSpMV_bufferSize(
			handleSparse,
			transA,
			&one,
			descrA,
			descrVecIn,
			&zero,
			descrVecOut,
			HIP_R_32F,
			HIPSPARSE_CSRMV_ALG1,
			&buffSize );
	char* buff;
	hipMalloc((void **) &buff, buffSize);
	//warm up
	for(int i = 0; i < 100; ++i){
		smpvStatus = 
		hipsparseSpMV(
			handleSparse,
			transA,
			&one,
			descrA,
			descrVecIn,
			&zero,
			descrVecOut,
			HIP_R_32F,
			HIPSPARSE_CSRMV_ALG1,
			buff);
	}
	gettimeofday(&start1, NULL);
	
    if(hipSuccess != hipMemcpy(vector_in_d, vector_in, dimension*sizeof(double), hipMemcpyHostToDevice)) printf("error4\n");
	while (iter<MAXIter){
		hipsparseStatus_t smpvStatus = 
		hipsparseSpMV(
			handleSparse,
			transA,
			&one,
			descrA,
			descrVecIn,
			&zero,
			descrVecOut,
			HIP_R_32F,
			HIPSPARSE_CSRMV_ALG1,
			buff);
		//hipsparseStatus_t smpvStatus = 
		//hipsparseDcsrmv(handleSparse,
		//		transA,
		//		dimension,
		//		dimension,
		//		totalNum,
		//		&one,
		//		descr,
		//		V_d,
		//		rowIdx_d,
		//		col_d,
		//		vector_in_d,
		//		&zero,
		//		vector_out_d);
		iter++;
	}
	hipMemcpy(vector_out, vector_out_d, dimension*sizeof(double), hipMemcpyDeviceToHost);
	gettimeofday(&end1, NULL);	
	double timeByMs=(double (end1.tv_sec * 1000000 + end1.tv_usec)-(start1.tv_sec * 1000000 + start1.tv_usec))/1000;
	printf("iter is %d, time is %f ms, GPU csrmv Gflops is %f\n ",iter, timeByMs, (1e-9*(totalNum*2)*1000*iter)/timeByMs);
			

}
void solverGPuUnprecondCUSPARSE(matrixCOO* localMatrix, 
		const double *vector_in, double *vector_out,  
		const int MAXIter)
{
	//exampine the performance using cusparse library functions with
	//CSR format
	//double dotp0,dotr0,dotr1,doth;
	int dimension, totalNum; 
    int *rowIdx, *J; 
    double* V;
    dimension = localMatrix->dimension; 
    totalNum = localMatrix->totalNum; 

	rowIdx = localMatrix->rowIdx; 
    J = localMatrix->J;
    V = localMatrix->V;
	
	int* col_d;
	int* rowIdx_d;
	double *V_d;

	double *vector_in_d, *vector_out_d;
	size_t size1=dimension*sizeof(double);
	
	hipMalloc((void **) &rowIdx_d, (dimension+1)*sizeof(double));
	hipMalloc((void **) &vector_out_d,size1);
	hipMalloc((void **) &vector_in_d,size1);
	hipMalloc((void **) &col_d,totalNum*sizeof(int));
	hipMalloc((void **) &V_d,totalNum*sizeof(double));
	//double *x=(double *) malloc(size1);
	int iter=0;
	//double const1 = 1.0;
	//initialize
   	if(hipSuccess != hipMemcpy(rowIdx_d, rowIdx, (dimension+1)*sizeof(int), hipMemcpyHostToDevice)) printf("error1\n");
    if(hipSuccess !=hipMemcpy(col_d, J, totalNum*sizeof(int), hipMemcpyHostToDevice)) printf("error2\n");
    if(hipSuccess !=hipMemcpy(V_d, V, totalNum*sizeof(double), hipMemcpyHostToDevice)) printf("error3\n");
	
	struct timeval start1, end1;
	
	//if BSR doing the format change 
	//cusparseStatus_tcusparseDcsr2gebsr_bufferSize(handle, dir, m, n, descrA, csrValA, csrRowPtrA, 
	//		csrColIndA, rowBlockDim, colBlockDim, pBufferSize);
	hipsparseHandle_t handleSparse;
	hipsparseCreate(&handleSparse);
	hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
	hipsparseMatDescr_t descr = 0;
	int status = hipsparseCreateMatDescr(&descr);
	if (status != HIPSPARSE_STATUS_SUCCESS ) {
		exit(0);	
	}
	double one = 1.0;
	double zero = 0.0;
	hipsparseSetMatType (descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase (descr, HIPSPARSE_INDEX_BASE_ZERO);
	size_t buffSize;
	hipsparseStatus_t smpvStatus = 
	cusparseCsrmvEx_bufferSize(handleSparse,
		//CUSPARSE_ALG_NAIVE,	
		CUSPARSE_ALG_MERGE_PATH,
		transA,
		dimension,
		dimension,
		totalNum,
		&one,
		HIP_R_32F,
		descr,
		V_d,
		HIP_R_32F,
		rowIdx_d,
		col_d,
		vector_in_d,
		HIP_R_32F,
		&zero,
		HIP_R_32F,
		vector_out_d,
		HIP_R_32F,
		HIP_R_32F,
		&buffSize );
	char* buff;
    hipMalloc((void **) &buff, buffSize);
	//warm up
	for(int i = 0; i < 100; ++i){
		hipsparseStatus_t smpvStatus = 
		cusparseCsrmvEx(handleSparse,
			//CUSPARSE_ALG_NAIVE,
			CUSPARSE_ALG_MERGE_PATH,
			transA,
			dimension,
			dimension,
			totalNum,
			&one,
			HIP_R_32F,
			descr,
			V_d,
			HIP_R_32F,
			rowIdx_d,
			col_d,
			vector_in_d,
			HIP_R_32F,
			&zero,
			HIP_R_32F,
			vector_out_d,
			HIP_R_32F,
			HIP_R_32F,
			buff);
	}
	gettimeofday(&start1, NULL);
	
    if(hipSuccess != hipMemcpy(vector_in_d, vector_in, dimension*sizeof(double), hipMemcpyHostToDevice)) printf("error4\n");
	while (iter<MAXIter){
		hipsparseStatus_t smpvStatus = 
		cusparseCsrmvEx(handleSparse,
			//CUSPARSE_ALG_NAIVE,
			CUSPARSE_ALG_MERGE_PATH,
			transA,
			dimension,
			dimension,
			totalNum,
			&one,
			HIP_R_32F,
			descr,
			V_d,
			HIP_R_32F,
			rowIdx_d,
			col_d,
			vector_in_d,
			HIP_R_32F,
			&zero,
			HIP_R_32F,
			vector_out_d,
			HIP_R_32F,
			HIP_R_32F,
			buff);
		//hipsparseStatus_t smpvStatus = 
		//hipsparseDcsrmv(handleSparse,
		//		transA,
		//		dimension,
		//		dimension,
		//		totalNum,
		//		&one,
		//		descr,
		//		V_d,
		//		rowIdx_d,
		//		col_d,
		//		vector_in_d,
		//		&zero,
		//		vector_out_d);
		iter++;
	}
	hipMemcpy(vector_out, vector_out_d, dimension*sizeof(double), hipMemcpyDeviceToHost);
	gettimeofday(&end1, NULL);	
	double timeByMs=(double (end1.tv_sec * 1000000 + end1.tv_usec)-(start1.tv_sec * 1000000 + start1.tv_usec))/1000;
	printf("iter is %d, time is %f ms, GPU csrmv Gflops is %f\n ",iter, timeByMs, (1e-9*(totalNum*2)*1000*iter)/timeByMs);
			

}


