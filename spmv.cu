#include "hip/hip_runtime.h"
#include "kernel.h"
#include "spmv.h"
#include "convert.h"
#include "reordering.h"

static void cudaMallocTransDataEHYB(matrixEHYB* localMatrix, matrixEHYB* localMatrix_d, 
		const int sizeBlockELL, const int sizeER){


	localMatrix_d->dimension = localMatrix->dimension;
	localMatrix_d->numOfRowER = localMatrix->numOfRowER;
	localMatrix_d->nParts = localMatrix->nParts;
	int blockNumER = ceil(((float) localMatrix->numOfRowER)/warpSize);

    hipMalloc((void **) &(localMatrix_d->biasVecBlockELL), localMatrix->nParts*blockPerPart*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->widthVecBlockELL), localMatrix->nParts*blockPerPart*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->partBoundary), (localMatrix->nParts+1)*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->valBlockELL), sizeBlockELL*sizeof(float));
    hipMalloc((void **) &(localMatrix_d->colBlockELL), sizeBlockELL*sizeof(int16_t));

    hipMalloc((void **) &(localMatrix_d->rowVecER), localMatrix_d->numOfRowER*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->biasVecER), blockNumER*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->widthVecER), blockNumER*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->colER), sizeER*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->valER), sizeER*sizeof(float));

    hipMemcpy(localMatrix_d->biasVecBlockELL, localMatrix->biasVecBlockELL, localMatrix->nParts*blockPerPart*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->widthVecBlockELL, localMatrix->widthVecBlockELL, localMatrix->nParts*blockPerPart*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->partBoundary, localMatrix->partBoundary, (localMatrix->nParts+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->valBlockELL, localMatrix->valBlockELL, sizeBlockELL*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->colBlockELL, localMatrix->colBlockELL, sizeBlockELL*sizeof(int16_t), hipMemcpyHostToDevice);

    hipMemcpy(localMatrix_d->rowVecER, localMatrix->rowVecER, localMatrix_d->numOfRowER*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->biasVecER, localMatrix->biasVecER, blockNumER*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->widthVecER, localMatrix->widthVecER, blockNumER*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->colER, localMatrix->colER, sizeER*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->valER, localMatrix->valER, sizeER*sizeof(float), hipMemcpyHostToDevice);
}
extern "C"
void spmvGPuEHYB(matrixCOO* localMatrix, 
		const float *vectorIn, float *vectorOut,  
		const int MAXIter, int *realIter)
{
	//This function treat y as input and x as output, (solve the equation Ax=y) y is the vector we already known, x is the vector we are looking for

	int sizeBlockELL, sizeER;
	int dimension = localMatrix->dimension;
	int totalNum = localMatrix->totalNum;
	matrixEHYB localMatrixEHYB, localMatrixEHYB_d;

	COO2EHYB(localMatrix, 
			&localMatrixEHYB,
			&sizeBlockELL,
			&sizeER);

	cudaMallocTransDataEHYB(&localMatrixEHYB,
			&localMatrixEHYB_d, 
			sizeBlockELL,
			sizeER);
	printf("sizeER is %d\n", sizeER);

	float *vectorIn_d, *vectorOut_d;
	int *biasIdxBlock_d;
	if(localMatrix->nParts <= 40){
		hipMalloc((void**) &biasIdxBlock_d, localMatrix->nParts*sizeof(int));
	}
	size_t size1 = dimension*sizeof(float);
	hipMalloc((void **) &vectorOut_d,size1);
	hipMalloc((void **) &vectorIn_d,size1);
	//float *x=(float *) malloc(size1);
	int iter=0;
	struct timeval start1, end1;

	//float *x=(float *) malloc(size1);
	//initialize
	//warm Up
	hipMemcpy(vectorIn_d, vectorIn, dimension*sizeof(float), hipMemcpyHostToDevice);
	for(int i = 0; i < 10; ++i){
		if(localMatrix->nParts <= 40)
		    matrixVectorEHYB_small(&localMatrixEHYB_d, localMatrix->kernelPerPart,  biasIdxBlock_d, vectorIn_d, vectorOut_d, -1);
		else
		    matrixVectorEHYB(&localMatrixEHYB_d, vectorIn_d, vectorOut_d, -1);

	}
	hipMemcpy(vectorOut, vectorOut_d, dimension*sizeof(float), hipMemcpyDeviceToHost);
	gettimeofday(&start1, NULL);
	hipMemcpy(vectorIn_d, vectorIn, dimension*sizeof(float), hipMemcpyHostToDevice);
	while (iter<MAXIter){
		//hipMemset(vectorOut_d, 0, dimension*sizeof(float));
		if(localMatrix->nParts <= 40)
		    matrixVectorEHYB_small(&localMatrixEHYB_d, localMatrix->kernelPerPart, biasIdxBlock_d, vectorIn_d, vectorOut_d, -1);
		else
		    matrixVectorEHYB(&localMatrixEHYB_d, vectorIn_d, vectorOut_d, -1);
		iter++;
	}
	hipMemcpy(vectorOut, vectorOut_d, dimension*sizeof(float), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	gettimeofday(&end1, NULL);	
	float timeByMs=((end1.tv_sec * 1000000 + end1.tv_usec)-(start1.tv_sec * 1000000 + start1.tv_usec))/1000;
	printf("iter is %d, time is %f ms, GPU Gflops is %f\n ",iter, timeByMs, 
			(1e-9*(totalNum*2)*1000*iter)/timeByMs );
	hipFree(localMatrixEHYB_d.valER);
	hipFree(localMatrixEHYB_d.colER);
	hipFree(localMatrixEHYB_d.biasVecER);
	hipFree(localMatrixEHYB_d.widthVecER);
	hipFree(localMatrixEHYB_d.rowVecER);
	hipFree(localMatrixEHYB_d.biasVecBlockELL);
	hipFree(localMatrixEHYB_d.widthVecBlockELL);
	hipFree(localMatrixEHYB_d.colBlockELL);
	hipFree(localMatrixEHYB_d.valBlockELL);
	hipFree(localMatrixEHYB_d.partBoundary);
}

void solverGPuUnprecondCUSPARSE(matrixCOO* localMatrix, 
		const float *vector_in, float *vector_out,  
		const int MAXIter)
{
	//exampine the performance using cusparse library functions with
	//CSR format
	//float dotp0,dotr0,dotr1,doth;
	int dimension, totalNum; 
    int *rowIdx, *J; 
    float* V;
    dimension = localMatrix->dimension; 
    totalNum = localMatrix->totalNum; 

	rowIdx = localMatrix->rowIdx; 
    J = localMatrix->J;
    V = localMatrix->V;
	
	int* col_d;
	int* rowIdx_d;
	float *V_d;

	float *vector_in_d, *vector_out_d;
	size_t size1=dimension*sizeof(float);
	
	hipMalloc((void **) &rowIdx_d, (dimension+1)*sizeof(float));
	hipMalloc((void **) &vector_out_d,size1);
	hipMalloc((void **) &vector_in_d,size1);
	hipMalloc((void **) &col_d,totalNum*sizeof(int));
	hipMalloc((void **) &V_d,totalNum*sizeof(float));
	//float *x=(float *) malloc(size1);
	int iter=0;
	//float const1 = 1.0;
	//initialize
   	if(hipSuccess != hipMemcpy(rowIdx_d, rowIdx, (dimension+1)*sizeof(int), hipMemcpyHostToDevice)) printf("error1\n");
    if(hipSuccess !=hipMemcpy(col_d, J, totalNum*sizeof(int), hipMemcpyHostToDevice)) printf("error2\n");
    if(hipSuccess !=hipMemcpy(V_d, V, totalNum*sizeof(float), hipMemcpyHostToDevice)) printf("error3\n");
	
	struct timeval start1, end1;
	
	//if BSR doing the format change 
	//cusparseStatus_tcusparseDcsr2gebsr_bufferSize(handle, dir, m, n, descrA, csrValA, csrRowPtrA, 
	//		csrColIndA, rowBlockDim, colBlockDim, pBufferSize);
	hipsparseHandle_t handleSparse;
	hipsparseCreate(&handleSparse);
	hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
	hipsparseMatDescr_t descr = 0;
	int status = hipsparseCreateMatDescr(&descr);
	if (status != HIPSPARSE_STATUS_SUCCESS ) {
		exit(0);	
	}
	float one = 1.0;
	float zero = 0.0;
	size_t buffSize;
	hipsparseSetMatType (descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase (descr, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseStatus_t smpvStatus = 
	cusparseCsrmvEx_bufferSize(handleSparse,
		//CUSPARSE_ALG_NAIVE,	
		CUSPARSE_ALG_MERGE_PATH,
		transA,
		dimension,
		dimension,
		totalNum,
		&one,
		HIP_R_32F,
		descr,
		V_d,
		HIP_R_32F,
		rowIdx_d,
		col_d,
		vector_in_d,
		HIP_R_32F,
		&zero,
		HIP_R_32F,
		vector_out_d,
		HIP_R_32F,
		HIP_R_32F,
		&buffSize );
	char* buff;
    hipMalloc((void **) &buff, buffSize);
	//warm up
	for(int i = 0; i < 100; ++i){
		hipsparseStatus_t smpvStatus = 
		cusparseCsrmvEx(handleSparse,
			//CUSPARSE_ALG_NAIVE,
			CUSPARSE_ALG_MERGE_PATH,
			transA,
			dimension,
			dimension,
			totalNum,
			&one,
			HIP_R_32F,
			descr,
			V_d,
			HIP_R_32F,
			rowIdx_d,
			col_d,
			vector_in_d,
			HIP_R_32F,
			&zero,
			HIP_R_32F,
			vector_out_d,
			HIP_R_32F,
			HIP_R_32F,
			buff);
	}
	gettimeofday(&start1, NULL);
	
    if(hipSuccess != hipMemcpy(vector_in_d, vector_in, dimension*sizeof(float), hipMemcpyHostToDevice)) printf("error4\n");
	while (iter<MAXIter){
		hipsparseStatus_t smpvStatus = 
		cusparseCsrmvEx(handleSparse,
			//CUSPARSE_ALG_NAIVE,
			CUSPARSE_ALG_MERGE_PATH,
			transA,
			dimension,
			dimension,
			totalNum,
			&one,
			HIP_R_32F,
			descr,
			V_d,
			HIP_R_32F,
			rowIdx_d,
			col_d,
			vector_in_d,
			HIP_R_32F,
			&zero,
			HIP_R_32F,
			vector_out_d,
			HIP_R_32F,
			HIP_R_32F,
			buff);
		//hipsparseStatus_t smpvStatus = 
		//hipsparseDcsrmv(handleSparse,
		//		transA,
		//		dimension,
		//		dimension,
		//		totalNum,
		//		&one,
		//		descr,
		//		V_d,
		//		rowIdx_d,
		//		col_d,
		//		vector_in_d,
		//		&zero,
		//		vector_out_d);
		iter++;
	}
	hipMemcpy(vector_out, vector_out_d, dimension*sizeof(float), hipMemcpyDeviceToHost);
	gettimeofday(&end1, NULL);	
	float timeByMs=((end1.tv_sec * 1000000 + end1.tv_usec)-(start1.tv_sec * 1000000 + start1.tv_usec))/1000;
	printf("iter is %d, time is %f ms, GPU csrmv Gflops is %f\n ",iter, timeByMs, (1e-9*(totalNum*2)*1000*iter)/timeByMs);
			

}

void spmvHYB(matrixCOO* localMatrix, 
		const float *vector_in, float *vector_out,  
		const int MAXIter)
{
	//exampine the performance using cusparse library functions with
	//CSR format
	//float dotp0,dotr0,dotr1,doth;
	int dimension, totalNum; 
    int *rowIdx, *J; 
    float* V;
    dimension = localMatrix->dimension; 
    totalNum = localMatrix->totalNum; 

	rowIdx = localMatrix->rowIdx; 
    J = localMatrix->J;
    V = localMatrix->V;
	
	int* col_d;
	int* rowIdx_d;
	float *V_d;

	float *vector_in_d, *vector_out_d;
	size_t size1=dimension*sizeof(float);
	
	hipMalloc((void **) &rowIdx_d, (dimension+1)*sizeof(float));
	hipMalloc((void **) &vector_out_d,size1);
	hipMalloc((void **) &vector_in_d,size1);
	hipMalloc((void **) &col_d,totalNum*sizeof(int));
	hipMalloc((void **) &V_d,totalNum*sizeof(float));
	//float *x=(float *) malloc(size1);
	int iter=0;
	//float const1 = 1.0;
	//initialize
   	if(hipSuccess != hipMemcpy(rowIdx_d, rowIdx, (dimension+1)*sizeof(int), hipMemcpyHostToDevice)) printf("error1\n");
    if(hipSuccess !=hipMemcpy(col_d, J, totalNum*sizeof(int), hipMemcpyHostToDevice)) printf("error2\n");
    if(hipSuccess !=hipMemcpy(V_d, V, totalNum*sizeof(float), hipMemcpyHostToDevice)) printf("error3\n");
	
	struct timeval start1, end1;
	
	//if BSR doing the format change 
	//cusparseStatus_tcusparseDcsr2gebsr_bufferSize(handle, dir, m, n, descrA, csrValA, csrRowPtrA, 
	//		csrColIndA, rowBlockDim, colBlockDim, pBufferSize);
	hipsparseHandle_t handleSparse;
	hipsparseCreate(&handleSparse);
	hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
	hipsparseMatDescr_t descrA;
	int status = hipsparseCreateMatDescr(&descrA);
	if (status != HIPSPARSE_STATUS_SUCCESS ) {
		exit(0);	
	}
	float one = 1.0;
	float zero = 0.0;
	size_t buffSize;
	hipsparseSetMatType (descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase (descrA, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseHybMat_t hybA;
	hipsparseCreateHybMat(&hybA);
	hipsparseStatus_t smpvStatus = 
	hipsparseScsr2hyb(handleSparse,
		dimension,
		dimension,
		descrA,
		V_d,
		rowIdx_d,
		col_d,
		hybA,
		1,
		HIPSPARSE_HYB_PARTITION_AUTO);	
		//warm up
	for(int i = 0; i < 100; ++i){
		hipsparseStatus_t smpvStatus = 
		hipsparseShybmv(handleSparse,
			transA,
			&one,
			descrA,
			hybA,
			vector_in_d,
			&zero,
			vector_out_d);
	}
	gettimeofday(&start1, NULL);
	
    if(hipSuccess != hipMemcpy(vector_in_d, vector_in, dimension*sizeof(float), hipMemcpyHostToDevice)) printf("error4\n");
	while (iter<MAXIter){
		int errorIdx = 0;
		float compareError;
		hipsparseStatus_t smpvStatus = 
		hipsparseShybmv(handleSparse,
			transA,
			&one,
			descrA,
			hybA,
			vector_in_d,
			&zero,
			vector_out_d);
		iter++;
	}
	hipMemcpy(vector_out, vector_out_d, dimension*sizeof(float), hipMemcpyDeviceToHost);
	gettimeofday(&end1, NULL);	
	float timeByMs=((end1.tv_sec * 1000000 + end1.tv_usec)-(start1.tv_sec * 1000000 + start1.tv_usec))/1000;
	printf("iter is %d, time is %f ms, GPU csrmv Gflops is %f\n ",iter, timeByMs, (1e-9*(totalNum*2)*1000*iter)/timeByMs);
			

}
