#include "hip/hip_runtime.h"
#include "kernel.h"
#include "spmv.h"
#include "convert.h"
#include "reordering.h"

static void cudaMallocTransDataEHYB(matrixEHYB* localMatrix, matrixEHYB* localMatrix_d, 
		const int sizeBlockELL, const int sizeER){


	localMatrix_d->dimension = localMatrix->dimension;
	localMatrix_d->numOfRowER = localMatrix->numOfRowER;
	localMatrix_d->nParts = localMatrix->nParts;
	int blockNumER = ceil(((float) localMatrix->numOfRowER)/warpSize);

    hipMalloc((void **) &(localMatrix_d->biasVecBlockELL), localMatrix->nParts*blockPerPart*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->widthVecBlockELL), localMatrix->nParts*blockPerPart*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->partBoundary), (localMatrix->nParts+1)*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->valBlockELL), sizeBlockELL*sizeof(float));
    hipMalloc((void **) &(localMatrix_d->colBlockELL), sizeBlockELL*sizeof(int16_t));

    hipMalloc((void **) &(localMatrix_d->rowVecER), localMatrix_d->numOfRowER*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->biasVecER), blockNumER*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->widthVecER), blockNumER*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->colER), sizeER*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->valER), sizeER*sizeof(float));

    hipMemcpy(localMatrix_d->biasVecBlockELL, localMatrix->biasVecBlockELL, localMatrix->nParts*blockPerPart*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->widthVecBlockELL, localMatrix->widthVecBlockELL, localMatrix->nParts*blockPerPart*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->partBoundary, localMatrix->partBoundary, (localMatrix->nParts+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->valBlockELL, localMatrix->valBlockELL, sizeBlockELL*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->colBlockELL, localMatrix->colBlockELL, sizeBlockELL*sizeof(int16_t), hipMemcpyHostToDevice);

    hipMemcpy(localMatrix_d->rowVecER, localMatrix->rowVecER, localMatrix_d->numOfRowER*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->biasVecER, localMatrix->biasVecER, blockNumER*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->widthVecER, localMatrix->widthVecER, blockNumER*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->colER, localMatrix->colER, sizeER*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->valER, localMatrix->valER, sizeER*sizeof(float), hipMemcpyHostToDevice);
}
extern "C"
void spmvGPuEHYB(matrixCOO* localMatrix, 
		const float *vectorIn, float *vectorOut,  
		const int MAXIter, int *realIter)
{
	//This function treat y as input and x as output, (solve the equation Ax=y) y is the vector we already known, x is the vector we are looking for

	int sizeBlockELL, sizeER;
	int dimension = localMatrix->dimension;
	int totalNum = localMatrix->totalNum;
	matrixEHYB localMatrixEHYB, localMatrixEHYB_d;

	COO2EHYB(localMatrix, 
			&localMatrixEHYB,
			&sizeBlockELL,
			&sizeER);

	cudaMallocTransDataEHYB(&localMatrixEHYB,
			&localMatrixEHYB_d, 
			sizeBlockELL,
			sizeER);
	printf("sizeER is %d\n", sizeER);

	float *vectorIn_d, *vectorOut_d;
	int16_t *biasIdxBlock_d;
	size_t size1 = dimension*sizeof(float);
	//hipMalloc((void **) &biasIdxBlock_d, sizeof(uint16_t)*localMatrix->nParts);
	hipMalloc((void **) &vectorOut_d,size1);
	hipMalloc((void **) &vectorIn_d,size1);
	//float *x=(float *) malloc(size1);
	int iter=0;
	struct timeval start1, end1;

	//float *x=(float *) malloc(size1);
	//initialize
	
	gettimeofday(&start1, NULL);
    hipMemcpy(vectorIn_d, vectorIn, dimension*sizeof(float), hipMemcpyHostToDevice);
	while (iter<MAXIter){
		matrixVectorEHYB(&localMatrixEHYB_d, vectorIn_d, vectorOut_d, -1);
		iter++;
	}
	hipMemcpy(vectorOut, vectorOut_d, dimension*sizeof(float), hipMemcpyDeviceToHost);
	gettimeofday(&end1, NULL);	
	float timeByMs=((end1.tv_sec * 1000000 + end1.tv_usec)-(start1.tv_sec * 1000000 + start1.tv_usec))/1000;
	printf("iter is %d, time is %f ms, GPU Gflops is %f\n ",iter, timeByMs, 
			(1e-9*(totalNum*2)*1000*iter)/timeByMs );
	hipFree(localMatrixEHYB_d.valER);
	hipFree(localMatrixEHYB_d.colER);
	hipFree(localMatrixEHYB_d.biasVecER);
	hipFree(localMatrixEHYB_d.widthVecER);
	hipFree(localMatrixEHYB_d.rowVecER);
	hipFree(localMatrixEHYB_d.biasVecBlockELL);
	hipFree(localMatrixEHYB_d.widthVecBlockELL);
	hipFree(localMatrixEHYB_d.colBlockELL);
	hipFree(localMatrixEHYB_d.valBlockELL);
	hipFree(localMatrixEHYB_d.partBoundary);
}

void solverGPuUnprecondCUSPARSE(matrixCOO* localMatrix, 
		const float *vector_in, float *vector_out,  
		const int MAXIter)
{
	//exampine the performance using cusparse library functions with
	//CSR format
	//float dotp0,dotr0,dotr1,doth;
	int dimension, totalNum; 
    int *rowIdx, *J; 
    float* V;
    dimension = localMatrix->dimension; 
    totalNum = localMatrix->totalNum; 

	rowIdx = localMatrix->rowIdx; 
    J = localMatrix->J;
    V = localMatrix->V;
	
	int* col_d;
	int* rowIdx_d;
	float *V_d;

	float *vector_in_d, *vector_out_d;
	size_t size1=dimension*sizeof(float);
	
	hipMalloc((void **) &rowIdx_d, (dimension+1)*sizeof(float));
	hipMalloc((void **) &vector_out_d,size1);
	hipMalloc((void **) &vector_in_d,size1);
	hipMalloc((void **) &col_d,totalNum*sizeof(int));
	hipMalloc((void **) &V_d,totalNum*sizeof(float));
	//float *x=(float *) malloc(size1);
	int iter=0;
	float const1 = 1.0;
	//initialize
   	if(hipSuccess != hipMemcpy(rowIdx_d, rowIdx, (dimension+1)*sizeof(int), hipMemcpyHostToDevice)) printf("error1\n");
    if(hipSuccess !=hipMemcpy(col_d, J, totalNum*sizeof(int), hipMemcpyHostToDevice)) printf("error2\n");
    if(hipSuccess !=hipMemcpy(V_d, V, totalNum*sizeof(float), hipMemcpyHostToDevice)) printf("error3\n");
	
	struct timeval start1, end1;
	
	//if BSR doing the format change 
	//cusparseStatus_tcusparseDcsr2gebsr_bufferSize(handle, dir, m, n, descrA, csrValA, csrRowPtrA, 
	//		csrColIndA, rowBlockDim, colBlockDim, pBufferSize);
	hipsparseHandle_t handleSparse;
	hipsparseCreate(&handleSparse);
	hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
	hipsparseMatDescr_t descr = 0;
	int status = hipsparseCreateMatDescr(&descr);
	if (status != HIPSPARSE_STATUS_SUCCESS ) {
		exit(0);	
	}
	hipsparseSetMatType (descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase (descr, HIPSPARSE_INDEX_BASE_ZERO);
	gettimeofday(&start1, NULL);
	float one = 1.0;
	float zero = 0.0;
    if(hipSuccess != hipMemcpy(vector_in_d, vector_in, dimension*sizeof(float), hipMemcpyHostToDevice)) printf("error4\n");
	while (iter<MAXIter){
		//int errorIdx = 0;
		//float compareError;
		
		hipsparseStatus_t smpvStatus = 
		hipsparseScsrmv(handleSparse,
				transA,
				dimension,
				dimension,
				totalNum,
				&one,
				descr,
				V_d,
				rowIdx_d,
				col_d,
				vector_in_d,
				&zero,
				vector_out_d);
		iter++;
	}
	hipMemcpy(vector_out, vector_out_d, dimension*sizeof(float), hipMemcpyDeviceToHost);
	gettimeofday(&end1, NULL);	
	float timeByMs=((end1.tv_sec * 1000000 + end1.tv_usec)-(start1.tv_sec * 1000000 + start1.tv_usec))/1000;
	printf("iter is %d, time is %f ms, GPU csrmv Gflops is %f\n ",iter, timeByMs, (1e-9*(totalNum*2)*1000*iter)/timeByMs);
			

}

