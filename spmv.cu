#include "hip/hip_runtime.h"
#include "kernel.h"
#include "spmv.h"
#include "convert.h"
#include "reordering.h"

static void cudaMallocTransDataEHYB(matrixEHYB* localMatrix, matrixEHYB* localMatrix_d, 
		const int sizeBlockELL, const int sizeER){


	localMatrix_d->dimension = localMatrix->dimension;
	localMatrix_d->kernelPerPart = localMatrix->kernelPerPart;
	localMatrix_d->numOfRowER = localMatrix->numOfRowER;
	localMatrix_d->nParts = localMatrix->nParts;
	int blockNumER = ceil(((double) localMatrix->numOfRowER)/warpSize);
	int warpIdxER = 0;	

    hipMalloc((void **) &(localMatrix_d->warpIdxER_d), sizeof(int));
    hipMalloc((void **) &(localMatrix_d->outER), localMatrix_d->numOfRowER*sizeof(double));
    hipMalloc((void **) &(localMatrix_d->biasVecBlockELL), localMatrix->nParts*blockPerPart*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->widthVecBlockELL), localMatrix->nParts*blockPerPart*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->partBoundary), (localMatrix->nParts+1)*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->valBlockELL), sizeBlockELL*sizeof(double));
    hipMalloc((void **) &(localMatrix_d->colBlockELL), sizeBlockELL*sizeof(int16_t));

    hipMalloc((void **) &(localMatrix_d->rowVecER), localMatrix_d->numOfRowER*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->biasVecER), blockNumER*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->widthVecER), blockNumER*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->colER), sizeER*sizeof(int));
    hipMalloc((void **) &(localMatrix_d->valER), sizeER*sizeof(double));

    hipMemcpy(localMatrix_d->warpIdxER_d, &warpIdxER, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->biasVecBlockELL, localMatrix->biasVecBlockELL, localMatrix->nParts*blockPerPart*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->widthVecBlockELL, localMatrix->widthVecBlockELL, localMatrix->nParts*blockPerPart*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->partBoundary, localMatrix->partBoundary, (localMatrix->nParts+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->valBlockELL, localMatrix->valBlockELL, sizeBlockELL*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->colBlockELL, localMatrix->colBlockELL, sizeBlockELL*sizeof(int16_t), hipMemcpyHostToDevice);

    hipMemcpy(localMatrix_d->rowVecER, localMatrix->rowVecER, localMatrix_d->numOfRowER*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->biasVecER, localMatrix->biasVecER, blockNumER*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->widthVecER, localMatrix->widthVecER, blockNumER*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->colER, localMatrix->colER, sizeER*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(localMatrix_d->valER, localMatrix->valER, sizeER*sizeof(double), hipMemcpyHostToDevice);
}
extern "C"
void spmvGPuEHYB(matrixCOO* localMatrix, 
		const double *vectorIn, double *vectorOut,  
		const int MAXIter, int *realIter)
{
	//This function treat y as input and x as output, (solve the equation Ax=y) y is the vector we already known, x is the vector we are looking for

	int sizeBlockELL, sizeER;
	int dimension = localMatrix->dimension;
	int totalNum = localMatrix->totalNum;
	matrixEHYB localMatrixEHYB, localMatrixEHYB_d;

	COO2EHYB(localMatrix, 
			&localMatrixEHYB,
			&sizeBlockELL,
			&sizeER);

	cudaMallocTransDataEHYB(&localMatrixEHYB,
			&localMatrixEHYB_d, 
			sizeBlockELL,
			sizeER);
	printf("sizeER is %d\n", sizeER);

	double *vectorIn_d, *vectorOut_d;
	int *biasIdxBlock_d;
	if(localMatrix->nParts <= 40){
		hipMalloc((void**) &biasIdxBlock_d, localMatrix->nParts*sizeof(int));
	}
	size_t size1 = dimension*sizeof(double);
	hipMalloc((void **) &vectorOut_d,size1);
	hipMalloc((void **) &vectorIn_d,size1);
	//double *x=(double *) malloc(size1);
	int iter=0;
	struct timeval start1, end1;

	//double *x=(double *) malloc(size1);
	//initialize
	//warm Up
	hipMemcpy(vectorIn_d, vectorIn, dimension*sizeof(double), hipMemcpyHostToDevice);
	for(int i = 0; i < 10; ++i){
		if(localMatrix->nParts <= 40)
		    matrixVectorEHYB_small(&localMatrixEHYB_d, biasIdxBlock_d, vectorIn_d, vectorOut_d);
		else
		    matrixVectorEHYB(&localMatrixEHYB_d, vectorIn_d, vectorOut_d);

	}
	hipMemcpy(vectorOut, vectorOut_d, dimension*sizeof(double), hipMemcpyDeviceToHost);
	gettimeofday(&start1, NULL);
	hipMemcpy(vectorIn_d, vectorIn, dimension*sizeof(double), hipMemcpyHostToDevice);
	while (iter<MAXIter){
		if(localMatrix->nParts <= 40)
		    matrixVectorEHYB_small(&localMatrixEHYB_d, biasIdxBlock_d, vectorIn_d, vectorOut_d);
		else
		    matrixVectorEHYB(&localMatrixEHYB_d, vectorIn_d, vectorOut_d);
		iter++;
	}
	hipMemcpy(vectorOut, vectorOut_d, dimension*sizeof(double), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	gettimeofday(&end1, NULL);	
	double timeByMs=(double (end1.tv_sec * 1000000 + end1.tv_usec)-(start1.tv_sec * 1000000 + start1.tv_usec))/1000;
	printf("iter is %d, time is %f ms, GPU Gflops is %f\n ",iter, timeByMs, 
			(1e-9*(totalNum*2)*1000*iter)/timeByMs );
	hipFree(localMatrixEHYB_d.valER);
	hipFree(localMatrixEHYB_d.colER);
	hipFree(localMatrixEHYB_d.biasVecER);
	hipFree(localMatrixEHYB_d.widthVecER);
	hipFree(localMatrixEHYB_d.rowVecER);
	hipFree(localMatrixEHYB_d.biasVecBlockELL);
	hipFree(localMatrixEHYB_d.widthVecBlockELL);
	hipFree(localMatrixEHYB_d.colBlockELL);
	hipFree(localMatrixEHYB_d.valBlockELL);
	hipFree(localMatrixEHYB_d.partBoundary);
}

void solverGPuUnprecondCUSPARSE(matrixCOO* localMatrix, 
		const double *vector_in, double *vector_out,  
		const int MAXIter)
{
	//exampine the performance using cusparse library functions with
	//CSR format
	//double dotp0,dotr0,dotr1,doth;
	int dimension, totalNum; 
    int *rowIdx, *J; 
    double* V;
    dimension = localMatrix->dimension; 
    totalNum = localMatrix->totalNum; 

	rowIdx = localMatrix->rowIdx; 
    J = localMatrix->J;
    V = localMatrix->V;
	
	int* col_d;
	int* rowIdx_d;
	double *V_d;

	double *vector_in_d, *vector_out_d;
	size_t size1=dimension*sizeof(double);
	
	hipMalloc((void **) &rowIdx_d, (dimension+1)*sizeof(double));
	hipMalloc((void **) &vector_out_d,size1);
	hipMalloc((void **) &vector_in_d,size1);
	hipMalloc((void **) &col_d,totalNum*sizeof(int));
	hipMalloc((void **) &V_d,totalNum*sizeof(double));
	//double *x=(double *) malloc(size1);
	int iter=0;
	//double const1 = 1.0;
	//initialize
   	if(hipSuccess != hipMemcpy(rowIdx_d, rowIdx, (dimension+1)*sizeof(int), hipMemcpyHostToDevice)) printf("error1\n");
    if(hipSuccess !=hipMemcpy(col_d, J, totalNum*sizeof(int), hipMemcpyHostToDevice)) printf("error2\n");
    if(hipSuccess !=hipMemcpy(V_d, V, totalNum*sizeof(double), hipMemcpyHostToDevice)) printf("error3\n");
	
	struct timeval start1, end1;
	
	//if BSR doing the format change 
	//cusparseStatus_tcusparseDcsr2gebsr_bufferSize(handle, dir, m, n, descrA, csrValA, csrRowPtrA, 
	//		csrColIndA, rowBlockDim, colBlockDim, pBufferSize);
	hipsparseHandle_t handleSparse;
	hipsparseCreate(&handleSparse);
	hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
	hipsparseMatDescr_t descr = 0;
	int status = hipsparseCreateMatDescr(&descr);
	if (status != HIPSPARSE_STATUS_SUCCESS ) {
		exit(0);	
	}
	double one = 1.0;
	double zero = 0.0;
	hipsparseSetMatType (descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase (descr, HIPSPARSE_INDEX_BASE_ZERO);
	size_t buffSize;
	hipsparseStatus_t smpvStatus = 
	cusparseCsrmvEx_bufferSize(handleSparse,
		//CUSPARSE_ALG_NAIVE,	
		CUSPARSE_ALG_MERGE_PATH,
		transA,
		dimension,
		dimension,
		totalNum,
		&one,
		HIP_R_64F,
		descr,
		V_d,
		HIP_R_64F,
		rowIdx_d,
		col_d,
		vector_in_d,
		HIP_R_64F,
		&zero,
		HIP_R_64F,
		vector_out_d,
		HIP_R_64F,
		HIP_R_64F,
		&buffSize );
	char* buff;
    hipMalloc((void **) &buff, buffSize);
	//warm up
	for(int i = 0; i < 100; ++i){
		hipsparseStatus_t smpvStatus = 
		cusparseCsrmvEx(handleSparse,
			//CUSPARSE_ALG_NAIVE,
			CUSPARSE_ALG_MERGE_PATH,
			transA,
			dimension,
			dimension,
			totalNum,
			&one,
			HIP_R_64F,
			descr,
			V_d,
			HIP_R_64F,
			rowIdx_d,
			col_d,
			vector_in_d,
			HIP_R_64F,
			&zero,
			HIP_R_64F,
			vector_out_d,
			HIP_R_64F,
			HIP_R_64F,
			buff);
	}
	gettimeofday(&start1, NULL);
	
    if(hipSuccess != hipMemcpy(vector_in_d, vector_in, dimension*sizeof(double), hipMemcpyHostToDevice)) printf("error4\n");
	while (iter<MAXIter){
		hipsparseStatus_t smpvStatus = 
		cusparseCsrmvEx(handleSparse,
			//CUSPARSE_ALG_NAIVE,
			CUSPARSE_ALG_MERGE_PATH,
			transA,
			dimension,
			dimension,
			totalNum,
			&one,
			HIP_R_64F,
			descr,
			V_d,
			HIP_R_64F,
			rowIdx_d,
			col_d,
			vector_in_d,
			HIP_R_64F,
			&zero,
			HIP_R_64F,
			vector_out_d,
			HIP_R_64F,
			HIP_R_64F,
			buff);
		//hipsparseStatus_t smpvStatus = 
		//hipsparseDcsrmv(handleSparse,
		//		transA,
		//		dimension,
		//		dimension,
		//		totalNum,
		//		&one,
		//		descr,
		//		V_d,
		//		rowIdx_d,
		//		col_d,
		//		vector_in_d,
		//		&zero,
		//		vector_out_d);
		iter++;
	}
	hipMemcpy(vector_out, vector_out_d, dimension*sizeof(double), hipMemcpyDeviceToHost);
	gettimeofday(&end1, NULL);	
	double timeByMs=(double (end1.tv_sec * 1000000 + end1.tv_usec)-(start1.tv_sec * 1000000 + start1.tv_usec))/1000;
	printf("iter is %d, time is %f ms, GPU csrmv Gflops is %f\n ",iter, timeByMs, (1e-9*(totalNum*2)*1000*iter)/timeByMs);
			

}

void spmvHYB(matrixCOO* localMatrix, 
		const double *vector_in, double *vector_out,  
		const int MAXIter)
{
	//exampine the performance using cusparse library functions with
	//CSR format
	//double dotp0,dotr0,dotr1,doth;
	int dimension, totalNum; 
    int *rowIdx, *J; 
    double* V;
    dimension = localMatrix->dimension; 
    totalNum = localMatrix->totalNum; 

	rowIdx = localMatrix->rowIdx; 
    J = localMatrix->J;
    V = localMatrix->V;
	
	int* col_d;
	int* rowIdx_d;
	double *V_d;

	double *vector_in_d, *vector_out_d;
	size_t size1=dimension*sizeof(double);
	
	hipMalloc((void **) &rowIdx_d, (dimension+1)*sizeof(double));
	hipMalloc((void **) &vector_out_d,size1);
	hipMalloc((void **) &vector_in_d,size1);
	hipMalloc((void **) &col_d,totalNum*sizeof(int));
	hipMalloc((void **) &V_d,totalNum*sizeof(double));
	//double *x=(double *) malloc(size1);
	int iter=0;
	//double const1 = 1.0;
	//initialize
   	if(hipSuccess != hipMemcpy(rowIdx_d, rowIdx, (dimension+1)*sizeof(int), hipMemcpyHostToDevice)) printf("error1\n");
    if(hipSuccess !=hipMemcpy(col_d, J, totalNum*sizeof(int), hipMemcpyHostToDevice)) printf("error2\n");
    if(hipSuccess !=hipMemcpy(V_d, V, totalNum*sizeof(double), hipMemcpyHostToDevice)) printf("error3\n");
	
	struct timeval start1, end1;
	
	//if BSR doing the format change 
	//cusparseStatus_tcusparseDcsr2gebsr_bufferSize(handle, dir, m, n, descrA, csrValA, csrRowPtrA, 
	//		csrColIndA, rowBlockDim, colBlockDim, pBufferSize);
	hipsparseHandle_t handleSparse;
	hipsparseCreate(&handleSparse);
	hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
	hipsparseMatDescr_t descrA;
	int status = hipsparseCreateMatDescr(&descrA);
	if (status != HIPSPARSE_STATUS_SUCCESS ) {
		exit(0);	
	}
	double one = 1.0;
	double zero = 0.0;
	hipsparseSetMatType (descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase (descrA, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseHybMat_t hybA;
	hipsparseCreateHybMat(&hybA);
	hipsparseStatus_t smpvStatus = 
	hipsparseDcsr2hyb(handleSparse,
		dimension,
		dimension,
		descrA,
		V_d,
		rowIdx_d,
		col_d,
		hybA,
		1,
		HIPSPARSE_HYB_PARTITION_AUTO);	
		//warm up
	for(int i = 0; i < 100; ++i){
		hipsparseStatus_t smpvStatus = 
		hipsparseDhybmv(handleSparse,
			transA,
			&one,
			descrA,
			hybA,
			vector_in_d,
			&zero,
			vector_out_d);
	}
	gettimeofday(&start1, NULL);
	
    if(hipSuccess != hipMemcpy(vector_in_d, vector_in, dimension*sizeof(double), hipMemcpyHostToDevice)) printf("error4\n");
	while (iter<MAXIter){
		hipsparseStatus_t smpvStatus = 
		hipsparseDhybmv(handleSparse,
			transA,
			&one,
			descrA,
			hybA,
			vector_in_d,
			&zero,
			vector_out_d);
		iter++;
	}
	hipMemcpy(vector_out, vector_out_d, dimension*sizeof(double), hipMemcpyDeviceToHost);
	gettimeofday(&end1, NULL);	
	double timeByMs=((end1.tv_sec * 1000000 + end1.tv_usec)-(start1.tv_sec * 1000000 + start1.tv_usec))/1000;
	printf("iter is %d, time is %f ms, GPU csrmv Gflops is %f\n ",iter, timeByMs, (1e-9*(totalNum*2)*1000*iter)/timeByMs);
			

}
